#include "hip/hip_runtime.h"

#include "update_params_3d.h"
#include <math.h>
#define THREADS_PER_BLOCK 512


/***********************************************

          Compute Cluster Parameters

************************************************/


__host__ void update_params(spix_params* aos_params, spix_helper* sp_helper, PointCloudData& data, SuperpixelParams3d& soa_params, SpixMetaData& args, Logger* logger){
 

    // -- launch parameters --
    int nspix_buffer = soa_params.nspix_sum * args.nspix_buffer_mult;

    int NumThreads = THREADS_PER_BLOCK;
    int vertex_nblocks = ceil( double(data.V) / double(THREADS_PER_BLOCK) ); 
    dim3 VertexBlocks(vertex_nblocks);

	int spix_nblocks = ceil( double(nspix_buffer) / double(THREADS_PER_BLOCK) );
    dim3 SpixBlocks(spix_nblocks);

    // -- clear all but the "valid" bool --
    clear_fields<<<SpixBlocks,NumThreads>>>(aos_params,sp_helper,nspix_buffer);
    hipMemset(sp_helper, 0, nspix_buffer*sizeof(spix_helper));

    // -- accumulate via sum --
    sum_by_label<<<VertexBlocks,NumThreads>>>(data.ftrs,data.pos,data.ptr,data.bids,
                                              soa_params.spix_ptr(),soa_params.csum_nspix_ptr(),
                                              aos_params,sp_helper,data.V,nspix_buffer);

    // -- compute sample stats --
    update_params_kernel<<<SpixBlocks,NumThreads>>>(aos_params,sp_helper,args.sigma2_app, args.sp_size, nspix_buffer);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

}


__global__
void clear_fields(spix_params* sp_params, spix_helper* sp_helper, const int nspix_total){

	int k = threadIdx.x + blockIdx.x * blockDim.x;  // the label
	if (k>=nspix_total) return;

	sp_params[k].count = 0;
    sp_params[k].logdet_sigma_shape = 0;

	float3 mu_app;
	mu_app.x = 0;
	mu_app.y = 0;
	mu_app.z = 0;
	sp_params[k].mu_app = mu_app;

	double3 mu_pos;
	mu_pos.x = 0;
	mu_pos.y = 0;
	mu_pos.z = 0;
	sp_params[k].mu_pos = mu_pos;

	double3 var_pos;
	var_pos.x = 0;
	var_pos.y = 0;
	var_pos.z = 0;
	sp_params[k].var_pos = var_pos;

	double3 cov_pos;
	cov_pos.x = 0;
	cov_pos.y = 0;
	cov_pos.z = 0;
	sp_params[k].cov_pos = cov_pos;
}

__global__
void sum_by_label(const float3* ftrs, const float3* pos,
                  const int* vptr, const uint8_t* vbids,
                  const uint32_t* spix, const uint32_t* csum_nspix,
                  spix_params* sp_params, spix_helper* sp_helper,
                  const int V_total, const int nspix_buffer) {

    // -- get vertex --
    int vertex = threadIdx.x + blockIdx.x * blockDim.x;
    if (vertex>=V_total) return;
    int bx = vbids[vertex];
    int V = vptr[bx+1] - vptr[bx];
    uint32_t spix_offset = csum_nspix[bx];

    // -- read superpixel label and some checks --
    uint32_t spix_id = spix[vertex];
    if (spix_id < 0){
        printf("invalid superpixel id.\n");
    }
    assert(spix_id >= 0);

    int spix_index = spix_id + spix_offset;
    if (sp_params[spix_index].valid != 1){
      printf("invalid but living spix[(%d,%d)]: %d %d\n",bx,vertex,spix_id,spix_index);
    }
    assert(sp_params[spix_index].valid==1);

    // -- unpack data --
    const float3 v_ftr = ftrs[vertex];
    const float3 v_pos = pos[vertex];

    // -- write features --
	atomicAdd(&sp_params[spix_index].count, 1);
    atomicAdd(&sp_helper[spix_index].sum_app.x, v_ftr.x);
    atomicAdd(&sp_helper[spix_index].sum_app.y, v_ftr.y);
    atomicAdd(&sp_helper[spix_index].sum_app.z, v_ftr.z);

    // -- write positions --
	atomicAdd(&sp_helper[spix_index].sum_pos.x, v_pos.x);
	atomicAdd(&sp_helper[spix_index].sum_pos.y, v_pos.y);
	atomicAdd(&sp_helper[spix_index].sum_pos.z, v_pos.z);

    atomicAdd(&sp_helper[spix_index].sq_sum_self_pos.x, v_pos.x*v_pos.x);
	atomicAdd(&sp_helper[spix_index].sq_sum_self_pos.y, v_pos.y*v_pos.y);
	atomicAdd(&sp_helper[spix_index].sq_sum_self_pos.z, v_pos.z*v_pos.z);

    atomicAdd(&sp_helper[spix_index].sq_sum_pairs_pos.x, v_pos.x*v_pos.y);
	atomicAdd(&sp_helper[spix_index].sq_sum_pairs_pos.y, v_pos.x*v_pos.z);
	atomicAdd(&sp_helper[spix_index].sq_sum_pairs_pos.z, v_pos.y*v_pos.z);
	
}



__device__ bool 
compute_and_invert_covariance_3d(double3 mu_pos, double3 sq_sum_self_pos, double3 sq_sum_pairs_pos,
                                 double prior_diag, double total_count, int count, double3& ivar, double3& icov, double& det) {
    
    // Check
    det = 0.001;
    if (total_count <= 3) return false;

    // Compute covariance matrix elements (local registers)
    float sxx = (prior_diag + sq_sum_self_pos.x - count*mu_pos.x*mu_pos.x) / (total_count - 3.0);  // Cov(x,x) sxx
    float sxy = (sq_sum_pairs_pos.x - count*mu_pos.x*mu_pos.y) / (total_count - 3.0);  // Cov(x,y) sxy
    float sxz = (sq_sum_pairs_pos.y - count*mu_pos.x*mu_pos.z) / (total_count - 3.0);;  // Cov(x,z) sxz
    float syy = (prior_diag + sq_sum_self_pos.y - count*mu_pos.y*mu_pos.y) / (total_count - 3.0);;  // Cov(y,y) syy
    float syz = (sq_sum_pairs_pos.z - count*mu_pos.y*mu_pos.z) / (total_count - 3.0);;  // Cov(y,z) syz
    float szz = (prior_diag + sq_sum_self_pos.z - count*mu_pos.z*mu_pos.z) / (total_count - 3.0);;  // Cov(z,z) szz
        
    // In your covariance function, before computing determinant:
    // printf("Raw variances: sxx=%.12f, syy=%.12f, szz=%.12f\n", sxx, syy, szz);
    // printf("Covariances: sxy=%.12f, sxz=%.12f, syz=%.12f\n", sxy, sxz, syz);

    // Compute determinant for inverse
    det = sxx * (syy * szz - syz * syz) 
        - sxy * (sxy * szz - sxz * syz) 
        + sxz * (sxy * syz - sxz * syy);

    // Check for singularity
    if (det < 1e-8f) {
        det = 0.01;
        return false;  // Matrix is singular
    }
    
    float inv_det = 1.0f / det;
    
    // Compute inverse matrix elements (adjugate / determinant)
    ivar.x = (syy * szz - syz * syz) * inv_det;  // [0,0]
    ivar.y = (sxx * szz - sxz * sxz) * inv_det;  // [1,1]  
    ivar.z = (sxx * syy - sxy * sxy) * inv_det;  // [2,2]
    icov.x = (sxz * syz - sxy * szz) * inv_det;  // [0,1]
    icov.y = (sxy * syz - sxz * syy) * inv_det;  // [0,2]
    icov.z = (sxy * sxz - sxx * syz) * inv_det;  // [1,2]
    return true;  // Success
}

__global__
void update_params_kernel(spix_params* sp_params, spix_helper* sp_helper,
                          float sigma_app, const int sp_size, const int nsuperpixel_buffer) {

    // -- update thread --
	int k = threadIdx.x + blockIdx.x * blockDim.x; // the label
	if (k>=nsuperpixel_buffer) return;
	if (sp_params[k].valid == 0) return;

    // -- unpack --
    spix_params p = sp_params[k];
    spix_helper h = sp_helper[k];
    if (p.count<=0){
      p.valid = 0;
      return;
    }
    
    // -- local vars --
    int pc = sp_size * sp_size;
    double prior_diag = pc*pc;
    double total_count = (double) p.count + pc*50;

    // -- compute means --
    float inv_n = 1./p.count;
    double3 sum_app = h.sum_app;
    float3 mu_app;
    mu_app.x = sum_app.x*inv_n;
    mu_app.y = sum_app.y*inv_n;
    mu_app.z = sum_app.z*inv_n;

    double3 sum_pos = h.sum_pos;
    double3 mu_pos;
    mu_pos.x = sum_pos.x*inv_n;
    mu_pos.y = sum_pos.y*inv_n;
    mu_pos.z = sum_pos.z*inv_n;

    // -- compute inverse cov --
    double3 ivar;
    double3 icov;
    double det;
    bool succ = compute_and_invert_covariance_3d(mu_pos,h.sq_sum_self_pos,h.sq_sum_pairs_pos,
                                                 prior_diag,total_count,p.count,ivar,icov,det);
    if (!succ) {
        icov.x = icov.y = icov.z = 0.0f;
        ivar.x = ivar.y = ivar.z = 1e2f;  // Large identity for inverse
    }

    // -- write all at once --
    sp_params[k].mu_app = mu_app;
    sp_params[k].mu_pos = mu_pos;
    sp_params[k].var_pos = ivar;
    sp_params[k].cov_pos = icov;
    sp_params[k].logdet_sigma_shape = log(det);
}