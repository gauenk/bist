#include "hip/hip_runtime.h"
#pragma once

#include <string>
#include <fstream>
#include <iostream>
#include <filesystem>
#include <algorithm>
#include <cfloat>

#include "init_utils.h"

#include <vector>
#include <thrust/device_ptr.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "scannet_reader.h"

// -- read each scene --
std::tuple<float3*,float3*,float3*,int*,int,int>
read_scene(const std::vector<std::filesystem::path>& scene_files){

    // -- read sizes --
    int batchsize = scene_files.size();
    int* nnodes_cpu = (int*)malloc(batchsize * sizeof(int));
    int total_nodes = 0;
    
    // First pass: get sizes
    int _ix = 0;
    for (const auto& scene_file : scene_files) {
        nnodes_cpu[_ix] = get_vertex_count(scene_file);
        total_nodes += nnodes_cpu[_ix];
        _ix++;
    }
    printf("total nodes: %d\n",total_nodes);

    // Reserve space for all points
    std::vector<float> ftrs(total_nodes);
    std::vector<float> pos(total_nodes);
    std::vector<float> dim_sizes(6*batchsize);

    // Second pass: append data
    int _bx = 0;
    _ix = 0;
    for (const auto& scene_file : scene_files) {
        // -- .. --
        ScanNetScene scene;
        if(!scene.read_ply(scene_file)){
            exit(1);
        }

        // -- .. --
        memcpy(&ftrs[_ix], scene.ftr.data(), scene.size * sizeof(float));
        memcpy(&pos[_ix], scene.pos.data(), scene.size * sizeof(float));

        // -- .. --
        dim_sizes[6*_bx+0] = scene.xmin;
        dim_sizes[6*_bx+1] = scene.xmax;
        dim_sizes[6*_bx+2] = scene.ymin;
        dim_sizes[6*_bx+3] = scene.ymax;
        dim_sizes[6*_bx+4] = scene.zmin;
        dim_sizes[6*_bx+5] = scene.zmax;
        _ix += scene.size;
        _bx += 1;
    }

    // -- view --
    std::cout << "=== Scene Batch Info ===" << std::endl;
    std::cout << "Batch size: " << batchsize << std::endl;
    std::cout << "Total nodes: " << total_nodes << std::endl;

    // Print per-scene info
    int offset = 0;
    for (int i = 0; i < batchsize; ++i) {
        std::cout << "Scene " << i << ": " << nnodes_cpu[i] << " points" << std::endl;
        std::cout << "  Bounds: [" << dim_sizes[6*i+0] << ", " << dim_sizes[6*i+1] << "] "
                << "[" << dim_sizes[6*i+2] << ", " << dim_sizes[6*i+3] << "] "
                << "[" << dim_sizes[6*i+4] << ", " << dim_sizes[6*i+5] << "]" << std::endl;
    }

    // Print first few points
    std::cout << "First 3 points:" << std::endl;
    for (int i = 0; i < std::min(9, (int)pos.size()); i += 3) {
        std::cout << "  Pos: (" << pos[i] << ", " << pos[i+1] << ", " << pos[i+2] << ")" << std::endl;
        std::cout << "  RGB: (" << ftrs[i] << ", " << ftrs[i+1] << ", " << ftrs[i+2] << ")" << std::endl;
    }
    
    // -- copy --
    float3* ftrs_cu = (float3*)easy_allocate(total_nodes,sizeof(float3));
    float3* pos_cu = (float3*)easy_allocate(total_nodes,sizeof(float3));
    float3* dim_sizes_cu = (float3*)easy_allocate(2*batchsize,sizeof(float3));
    int* nnodes_cu = (int*)easy_allocate(batchsize,sizeof(int));
    hipDeviceSynchronize();
    hipMemcpy(ftrs_cu,thrust::raw_pointer_cast(ftrs.data()),total_nodes*sizeof(float3),hipMemcpyHostToDevice);
    hipMemcpy(pos_cu,thrust::raw_pointer_cast(pos.data()),total_nodes*sizeof(float3),hipMemcpyHostToDevice);
    hipMemcpy(dim_sizes_cu,thrust::raw_pointer_cast(dim_sizes.data()),2*batchsize*sizeof(float3),hipMemcpyHostToDevice);
    hipMemcpy(nnodes_cu,nnodes_cpu,batchsize*sizeof(int),hipMemcpyHostToDevice);
    return std::tuple(ftrs_cu,pos_cu,dim_sizes_cu,nnodes_cu,total_nodes,batchsize);

}

// -- write each scene; [nnodes == spix if point-cloud is the superpixel point cloud] --
bool write_scene(const std::vector<std::filesystem::path>& scene_files, 
                const std::filesystem::path& output_root, 
                float3* ftrs_cu, float3* pos_cu, int* nnodes_cu, int total_nodes, long* labels_cu){

    // -- allocate --
    int nbatch = scene_files.size();
    float* ftrs = (float*)malloc(3*total_nodes*sizeof(float));
    float* pos = (float*)malloc(3*total_nodes*sizeof(float));
    //float* dim_sizes = (float*)malloc(6*nbatch,sizeof(float));
    int* nnodes = (int*)malloc(nbatch*sizeof(int));
    long* labels = nullptr;
    if (labels_cu != nullptr){
        labels = (long*)malloc(total_nodes*sizeof(long));
    }

    // -- read to cpu --
    hipMemcpy(ftrs,ftrs_cu,total_nodes*sizeof(float3),hipMemcpyDeviceToHost);
    hipMemcpy(pos,pos_cu,total_nodes*sizeof(float3),hipMemcpyDeviceToHost);
    //hipMemcpy(dim_sizes,dim_sizes_cu,2*nbatch*sizeof(float3),hipMemcpyDeviceToHost);
    hipMemcpy(nnodes,nnodes_cu,nbatch*sizeof(int),hipMemcpyDeviceToHost);
    if (labels_cu != nullptr){
        hipMemcpy(labels,labels_cu,total_nodes*sizeof(long),hipMemcpyDeviceToHost);
    }

    // Second pass: append data
    float* ftrs_b = ftrs;
    float* pos_b = pos;
    int* nnodes_b = nnodes;
    for (const auto& scene_file : scene_files) {

        // -- .. --
        ScanNetScene scene;
        if(!scene.write_ply(scene_file,output_root,ftrs,pos,nnodes[0],labels)){
            exit(1);
        }

        // -- update --
        ftrs_b += 3*nnodes_b[0];
        pos_b += 3*nnodes_b[0];
        nnodes_b += 1;

    }


    free(ftrs);
    free(pos);
    free(nnodes);
    if (labels != nullptr){
        free(labels);
    }
    //free(dim_sizes);
    return 0;
}

int get_vertex_count(const std::filesystem::path& scene_path) {

    // -- get filenames --
    std::string scene_name = scene_path.filename().string();
    std::filesystem::path ply_file = scene_path / (scene_name + "_vh_clean_2.ply");
    std::ifstream file(ply_file.string());
    if (!file.is_open()) {
        printf("didn't get the vertex count!\n");
        return -1;
    }

    
    std::string line;
    while (std::getline(file, line)) {
        if (line.find("element vertex") != std::string::npos) {
            return std::stoi(line.substr(15));
        }
        if (line == "end_header") break;  // Stop if we hit end of header
    }
    return -1;
}

ScanNetScene::ScanNetScene() : size(0), 
    xmin(FLT_MAX), xmax(-FLT_MAX), 
    ymin(FLT_MAX), ymax(-FLT_MAX),
    zmin(FLT_MAX), zmax(-FLT_MAX) {}
    
bool ScanNetScene::read_ply(const std::filesystem::path& scene_path) {

    // -- helper --
    std::string line;

    // -- get filenames --
    std::string scene_name = scene_path.filename().string();
    std::filesystem::path info_file = scene_path / (scene_name + ".txt");
    std::filesystem::path ply_file = scene_path / (scene_name + "_vh_clean_2.ply");
    std::cout << info_file << std::endl;
    std::cout << ply_file << std::endl;

    // -- read the axis alignment matrix --
    std::ifstream info_stream(info_file);
    float axis_align[16];
    while (std::getline(info_stream, line)) {
        if (line.rfind("axisAlignment", 0) == 0) { // starts with "axisAlignment"
            std::istringstream iss(line.substr(line.find('=') + 1));
            for (int i = 0; i < 16; i++) {
                if (!(iss >> axis_align[i])) {
                    std::cerr << "Error parsing axisAlignment\n";
                    return 1;
                }
            }
            break;
        }
    }

    // // Print result to verify
    // for (int i = 0; i < 16; i++) {
    //     std::cout << axis_align[i] << (i % 4 == 3 ? "\n" : " ");
    // }

    // -- read ply file --
    std::cout << ply_file.string() << std::endl;
    std::ifstream file(ply_file.string(), std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Can not open: " << ply_file << std::endl;
        return false;
    }
    
    // Parse header
    int vertex_count = 0;
    bool binary_format = false;
    
    while (std::getline(file, line)) {
        std::cout << line << std::endl;
        if (line.find("element vertex") != std::string::npos) {
            vertex_count = std::stoi(line.substr(15));
        } else if (line.find("format binary") != std::string::npos) {
            binary_format = true;
        } else if (line == "end_header") {
            break;
        }
    }
    printf("vertex count: %d\n",vertex_count);

    // Allocate vectors
    pos.reserve(3*vertex_count);
    ftr.reserve(3*vertex_count);
    
    // Read data
    if (binary_format) {
        for (int i = 0; i < vertex_count; ++i) {
            float x, y, z;
            unsigned char r, g, b, alpha;
            
            // -- read --
            file.read(reinterpret_cast<char*>(&x), sizeof(float));
            file.read(reinterpret_cast<char*>(&y), sizeof(float));
            file.read(reinterpret_cast<char*>(&z), sizeof(float));
            file.read(reinterpret_cast<char*>(&r), sizeof(unsigned char));
            file.read(reinterpret_cast<char*>(&g), sizeof(unsigned char));
            file.read(reinterpret_cast<char*>(&b), sizeof(unsigned char));
            file.read(reinterpret_cast<char*>(&alpha), sizeof(unsigned char));

            // -- axis align --
            float x_new = axis_align[0]*x + axis_align[1]*y + axis_align[2]*z + axis_align[3];
            float y_new = axis_align[4]*x + axis_align[5]*y + axis_align[6]*z + axis_align[7];
            float z_new = axis_align[8]*x + axis_align[9]*y + axis_align[10]*z + axis_align[11];

            // -- update --
            x = x_new;
            y = y_new;
            z = z_new;

            // -- update bounds --
            xmin = std::min(xmin, x);
            xmax = std::max(xmax, x);
            ymin = std::min(ymin, y);
            ymax = std::max(ymax, y);
            zmin = std::min(zmin, z);
            zmax = std::max(zmax, z);
            
            // -- append --
            pos[3*i+0] = x;
            pos[3*i+1] = y;
            pos[3*i+2] = z;
            ftr[3*i+0] = r / 255.0f;
            ftr[3*i+1] = g / 255.0f;
            ftr[3*i+2] = b / 255.0f;
        }
    } else {
        for (int i = 0; i < vertex_count; ++i) {
            // -- read --
            float x, y, z;
            int r, g, b, alpha;
            file >> x >> y >> z >> r >> g >> b >> alpha;
            //printf("x,y,z: %.2f %.2f %.2f\n",x,y,z);
            
            // -- axis align --
            float x_new = axis_align[0]*x + axis_align[1]*y + axis_align[2]*z + axis_align[3];
            float y_new = axis_align[4]*x + axis_align[5]*y + axis_align[6]*z + axis_align[7];
            float z_new = axis_align[8]*x + axis_align[9]*y + axis_align[10]*z + axis_align[11];

            // -- update --
            x = x_new;
            y = y_new;
            z = z_new;

            // -- update bounds --
            xmin = std::min(xmin, x);
            xmax = std::max(xmax, x);
            ymin = std::min(ymin, y);
            ymax = std::max(ymax, y);
            zmin = std::min(zmin, z);
            zmax = std::max(zmax, z);

            // -- append --
            pos[3*i+0] = x;
            pos[3*i+1] = y;
            pos[3*i+2] = z;
            ftr[3*i+0] = r / 255.0f;
            ftr[3*i+1] = g / 255.0f;
            ftr[3*i+2] = b / 255.0f;
        }
    }
    
    size = vertex_count;
    file.close();
    return true;
};

bool ScanNetScene::write_ply(const std::filesystem::path& scene_path, 
                            const std::filesystem::path& output_root,
                            float* ftrs, float* pos, int nnodes, long* labels) {

    // -- helper --
    std::string line;

    // -- make dir if needed --
    std::string scene_name = scene_path.filename().string();
    std::filesystem::path write_path = output_root / scene_name;
    if (!std::filesystem::exists(write_path)) {
        std::filesystem::create_directories(write_path);
    }

    // -- get filenames --
    std::filesystem::path ply_file = write_path / (scene_name + "_vh_clean_2.ply");
    std::cout << ply_file << std::endl;
    
    // -- delete existing file if it exists --
    if (std::filesystem::exists(ply_file)) {
        std::filesystem::remove(ply_file);
    }

    // -- open file for writing --
    std::ofstream file(ply_file.string(), std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Can not open: " << ply_file.string() << std::endl;
        return false;
    }

    // -- write PLY header --
    std::string header = "ply\n";
    header += "format binary_little_endian 1.0\n";
    header += "comment MLIB generated\n";
    header += "element vertex " + std::to_string(nnodes) + "\n";
    header += "property float x\n";
    header += "property float y\n";
    header += "property float z\n";
    header += "property uchar red\n";
    header += "property uchar green\n";
    header += "property uchar blue\n";
    header += "property uchar alpha\n";
    if (labels != nullptr) {
        header += "property int label\n";
    }
    header += "end_header\n";
    file.write(header.c_str(), header.length());

    // -- write data --
    for (int i = 0; i < nnodes; ++i) {
        
        // -- init --
        float x, y, z;
        unsigned char r, g, b, alpha;

        // -- unpack --
        x = pos[3*i+0];
        y = pos[3*i+1];
        z = pos[3*i+2];
        r = static_cast<unsigned char>(ftrs[3*i+0] * 255.0f);
        g = static_cast<unsigned char>(ftrs[3*i+1] * 255.0f);
        b = static_cast<unsigned char>(ftrs[3*i+2] * 255.0f);
        alpha = 255;

        // -- write --
        file.write(reinterpret_cast<const char*>(&x), sizeof(float));
        file.write(reinterpret_cast<const char*>(&y), sizeof(float));
        file.write(reinterpret_cast<const char*>(&z), sizeof(float));
        file.write(reinterpret_cast<const char*>(&r), sizeof(unsigned char));
        file.write(reinterpret_cast<const char*>(&g), sizeof(unsigned char));
        file.write(reinterpret_cast<const char*>(&b), sizeof(unsigned char));
        file.write(reinterpret_cast<const char*>(&alpha), sizeof(unsigned char));
        // -- write label if provided --
        if (labels != nullptr) {
            file.write(reinterpret_cast<const char*>(&labels[i]), sizeof(long));
        }

    }
    
    file.close();
    return true;

};

// // Usage
// int main() {
//     ScanNetScene scene;
    
//     if (scene.load_from_ply("/path/to/scene0000_00_vh_clean_2.ply")) {
//         std::cout << "Loaded " << scene.size << " points" << std::endl;
//         std::cout << "Position vector size: " << scene.pos.size() << std::endl;
//         std::cout << "Feature vector size: " << scene.ftr.size() << std::endl;
        
//         // Now you can copy to CUDA:
//         // hipMemcpy(cuda_pos, scene.pos.data(), scene.size * sizeof(float3), hipMemcpyHostToDevice);
//         // hipMemcpy(cuda_ftr, scene.ftr.data(), scene.size * sizeof(float3), hipMemcpyHostToDevice);
//     }
    
//     return 0;
// }