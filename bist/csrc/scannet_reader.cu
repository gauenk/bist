#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <filesystem>
#include <algorithm>
#include <cfloat>

#include <vector>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>


#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"


#include "extract_edges.h"
#include "init_utils.h"
#include "scannet_reader.h"
#include "structs_3d.h"


std::vector<std::filesystem::path> get_scene_files(std::filesystem::path root) {
   std::vector<std::filesystem::path> scene_files;
   for (const auto& entry : std::filesystem::directory_iterator(root)) {
       if (entry.is_directory()) {
           scene_files.push_back(entry.path());
       }
   }
   return scene_files;
}


PointCloudData read_scene(const std::vector<std::filesystem::path>& scene_files) {
    
    // Initialize batch processing
    const int batch_size = scene_files.size();
    printf("Processing %d scene files\n", batch_size);

    // Initialize offset arrays for batching
    std::vector<int> vertex_ptr(batch_size + 1, 0);
    std::vector<int> edge_ptr(batch_size + 1, 0);
    std::vector<int> face_ptr(batch_size + 1, 0);
    
    // First pass: Calculate cumulative sizes for memory allocation
    for (int batch_idx = 0; batch_idx < batch_size; ++batch_idx) {
        const auto& scene_file = scene_files[batch_idx];
        vertex_ptr[batch_idx + 1] = vertex_ptr[batch_idx] + get_vertex_count(scene_file);
        face_ptr[batch_idx + 1] = face_ptr[batch_idx] + get_face_count(scene_file);
    }
    
    const int total_vertices = vertex_ptr[batch_size];
    const int total_faces = face_ptr[batch_size];
    printf("Total vertices: %d, Total faces: %d\n", total_vertices, total_faces);

    // Allocate host memory for all data
    std::vector<float3> features(total_vertices);
    std::vector<float3> positions(total_vertices);
    std::vector<float> bounding_boxes(6 * batch_size, 0.0f);  // xmin,xmax,ymin,ymax,zmin,zmax per scene
    std::vector<uint8_t> vertex_batch_ids(total_vertices, 0);
    std::vector<uint32_t> faces(3 * total_faces, 0);
    
    // Edge data (stored on device)
    thrust::device_vector<uint32_t> all_edges;
    std::vector<uint8_t> edge_batch_ids;

    // Second pass: Load and copy scene data
    int vertex_offset = 0;
    for (int batch_idx = 0; batch_idx < batch_size; ++batch_idx) {
        const auto& scene_file = scene_files[batch_idx];
        
        // Load scene data
        ScanNetScene scene;
        if (!scene.read_ply(scene_file)) {
            fprintf(stderr, "Failed to read scene file: %s\n", scene_file.c_str());
            exit(1);
        }

        // Copy vertex features and positions
        const int scene_vertex_count = scene.size;
        memcpy(&features[vertex_offset], scene.ftr.data(), scene_vertex_count * sizeof(float3));
        memcpy(&positions[vertex_offset], scene.pos.data(), scene_vertex_count * sizeof(float3));
        
        // Set batch IDs for this scene's vertices
        std::fill(vertex_batch_ids.begin() + vertex_offset, 
                  vertex_batch_ids.begin() + vertex_offset + scene_vertex_count, 
                  batch_idx);

        // Copy face data
        const int scene_face_count = face_ptr[batch_idx + 1] - face_ptr[batch_idx];
        const int face_offset = face_ptr[batch_idx];
        memcpy(&faces[3 * face_offset], scene.faces.data(), 3 * scene_face_count * sizeof(uint32_t));

        // Extract and append edges
        thrust::device_vector<uint32_t> scene_edges = extract_edges_from_pairs(scene.e0, scene.e1);
        const size_t scene_edge_count = scene_edges.size() / 2;  // Each edge has 2 vertices
        
        all_edges.insert(all_edges.end(), scene_edges.begin(), scene_edges.end());
        edge_batch_ids.insert(edge_batch_ids.end(), scene_edge_count, batch_idx);
        
        edge_ptr[batch_idx + 1] = edge_ptr[batch_idx] + scene_edge_count;
        printf("Scene %d edges: %d (cumulative: %d)\n", 
               batch_idx, (int)scene_edge_count, edge_ptr[batch_idx + 1]);

        // Store bounding box information
        const int bbox_offset = 6 * batch_idx;
        bounding_boxes[bbox_offset + 0] = scene.xmin;
        bounding_boxes[bbox_offset + 1] = scene.xmax;
        bounding_boxes[bbox_offset + 2] = scene.ymin;
        bounding_boxes[bbox_offset + 3] = scene.ymax;
        bounding_boxes[bbox_offset + 4] = scene.zmin;
        bounding_boxes[bbox_offset + 5] = scene.zmax;
        
        vertex_offset += scene_vertex_count;
    }

    // Print summary information
    // -- view --
    std::cout << "=== Scene Batch Info ===" << std::endl;
    std::cout << "Batch size: " << batch_size << std::endl;
    std::cout << "Total vertices: " << total_vertices << std::endl;

    // Print per-scene info
    for (int i = 0; i < batch_size; ++i) {
        std::cout << "Scene " << i << ": " << (vertex_ptr[i+1] - vertex_ptr[i]) << " points" << std::endl;
        std::cout << "  Bounds: [" << bounding_boxes[6*i+0] << ", " << bounding_boxes[6*i+1] << "] "
                << "[" << bounding_boxes[6*i+2] << ", " << bounding_boxes[6*i+3] << "] "
                << "[" << bounding_boxes[6*i+4] << ", " << bounding_boxes[6*i+5] << "]" << std::endl;
    }

    // Print first few points
    std::cout << "First 3 points:" << std::endl;
    for (int i = 0; i < std::min(3, total_vertices); i++) {
        std::cout << "  Pos: (" << positions[i].x << ", " << positions[i].y << ", " << positions[i].z << ")" << std::endl;
        std::cout << "  RGB: (" << features[i].x << ", " << features[i].y << ", " << features[i].z << ")" << std::endl;
    }

    // Create and return result
    const int total_edges = edge_ptr[batch_size];
    printf("Total edges: %d (edge batch IDs: %zu)\n", total_edges, edge_batch_ids.size());
    
    // Note: These appear to be unused in the original code
    thrust::device_vector<uint8_t> unused_gcolors;
    thrust::device_vector<uint32_t> unused_csr_edges;
    thrust::device_vector<uint32_t> unused_csr_eptr;

    return PointCloudData(
        features, positions, faces, all_edges,
        vertex_batch_ids, edge_batch_ids, vertex_ptr, edge_ptr, face_ptr,
        bounding_boxes, 0, batch_size, total_vertices, total_edges, total_faces
    );
}



// -- write each scene; [nnodes == spix if point-cloud is the superpixel point cloud] --
bool write_scene(const std::vector<std::filesystem::path>& scene_files, 
                const std::filesystem::path& output_root, PointCloudData& data){
                // float3* ftrs_cu, float3* pos_cu, uint32_t* edges_cu, int* ptr_cu, int* eptr_cu, 
                // uint8_t* gcolor_cu, uint32_t* labels_cu){

    // -- sync before io --
    hipDeviceSynchronize();

    // // -- read nnodes --
    // int nbatch = scene_files.size();
    // int nnodes;
    // hipMemcpy(&nnodes,&ptr_cu[nbatch],sizeof(int),hipMemcpyDeviceToHost);
    // int nedges;
    // hipMemcpy(&nedges,&eptr_cu[nbatch],sizeof(int),hipMemcpyDeviceToHost);
    // printf("nnodes: %d\n",nnodes);
    // printf("nedges: %d\n",nedges);
    
    // // -- allocate --
    // float* ftrs = (float*)malloc(3*nnodes*sizeof(float));
    // float* pos = (float*)malloc(3*nnodes*sizeof(float));
    // uint32_t* edges = (uint32_t*)malloc(2*nedges*sizeof(uint32_t));
    // //float* dim_sizes = (float*)malloc(6*nbatch,sizeof(float));
    // int* ptr = (int*)malloc((nbatch+1)*sizeof(int));
    // int* eptr = (int*)malloc((nbatch+1)*sizeof(int));
    // uint8_t* gcolor = (uint8_t*)malloc(nnodes*sizeof(uint8_t));
    // uint32_t* labels = nullptr;
    // if (labels_cu != nullptr){
    //     labels = (uint32_t*)malloc(nnodes*sizeof(uint32_t));
    // }

    // // -- read to cpu --
    // hipMemcpy(ftrs,ftrs_cu,nnodes*sizeof(float3),hipMemcpyDeviceToHost);
    // hipMemcpy(pos,pos_cu,nnodes*sizeof(float3),hipMemcpyDeviceToHost);
    // hipMemcpy(edges,edges_cu,2*nedges*sizeof(uint32_t),hipMemcpyDeviceToHost);
    // //hipMemcpy(dim_sizes,dim_sizes_cu,2*nbatch*sizeof(float3),hipMemcpyDeviceToHost);
    // hipMemcpy(ptr,ptr_cu,(nbatch+1)*sizeof(int),hipMemcpyDeviceToHost);
    // hipMemcpy(eptr,eptr_cu,(nbatch+1)*sizeof(int),hipMemcpyDeviceToHost);
    // hipMemcpy(gcolor,gcolor_cu,nnodes*sizeof(uint8_t),hipMemcpyDeviceToHost);
    // if (labels_cu != nullptr){
    //     hipMemcpy(labels,labels_cu,nnodes*sizeof(uint32_t),hipMemcpyDeviceToHost);
    // }
    // hipDeviceSynchronize();
    // Second pass: append data
    // float* ftrs_b = ftrs;
    // float* pos_b = pos;
    // int* ptr_b = ptr;
    for(int batch_index=0; batch_index < data.B; batch_index++){
    //for (const auto& scene_file : scene_files) {
        
        // // -- get pointers --
        // float3* ftrs_b = &ftrs[3*ptr[ix]];
        // float3* pos_b = &pos[3*ptr[ix]];
        // uint32_t* edges_b = &edges[2*eptr[ix]];
        // uint8_t*  gcolor_b = &gcolor[ptr[ix]];
        // uint32_t* labels_b = (labels != nullptr) ? &labels[ptr[ix]] : nullptr;
        // int nnodes = ptr[ix+1] - ptr[ix];
        // int nedges = eptr[ix+1] - eptr[ix];
        // printf("nedges: %d\n",nedges);
        // if (labels!=nullptr){
        //     printf("labels_b: %ld\n",labels_b[0]);
        // }

        // -- write original data (for dev) --
        auto& scene_file = scene_files[batch_index];
        ScanNetScene scene;
        PointCloudDataHost host_data(data,batch_index);
        for (int ix = 0; ix< 10; ix++){
            printf("ftrs: %2.2f %2.2f %2.2f\n",host_data.ftrs[ix].x,host_data.ftrs[ix].y,host_data.ftrs[ix].z);
        }
        // if(!scene.write_ply_with_fn(scene_file,output_root,ftrs_b,pos_b,edges_b,nnodes,nedges,gcolor_b,labels_b)){
        if(!scene.write_ply_with_fn(scene_file,output_root,host_data)){

            exit(1);
        }

        // ix += 1;
    }


    // free(ftrs);
    // free(pos);
    // free(edges);
    // free(ptr);
    // free(eptr);
    // free(gcolor);
    // if (labels != nullptr){
    //     free(labels);
    // }
    //free(dim_sizes);
    return 0;
}

int get_vertex_count(const std::filesystem::path& scene_path) {

    // -- get filenames --
    std::string scene_name = scene_path.filename().string();
    std::filesystem::path ply_file = scene_path / (scene_name + "_vh_clean_2.ply");
    std::ifstream file(ply_file.string());
    if (!file.is_open()) {
        printf("didn't get the vertex count!\n");
        return -1;
    }

    
    std::string line;
    while (std::getline(file, line)) {
        if (line.find("element vertex") != std::string::npos) {
            return std::stoi(line.substr(15));
        }
        if (line == "end_header") break;  // Stop if we hit end of header
    }
    return -1;
}

int get_face_count(const std::filesystem::path& scene_path) {

    // -- get filenames --
    std::string scene_name = scene_path.filename().string();
    std::filesystem::path ply_file = scene_path / (scene_name + "_vh_clean_2.ply");
    std::ifstream file(ply_file.string());
    if (!file.is_open()) {
        printf("didn't get the vertex count!\n");
        return -1;
    }

    
    std::string line;
    while (std::getline(file, line)) {
        if (line.find("element face") != std::string::npos) {
            return std::stoi(line.substr(13));
        }
        if (line == "end_header") break;  // Stop if we hit end of header
    }
    return -1;
}

ScanNetScene::ScanNetScene() : size(0), 
    xmin(FLT_MAX), xmax(-FLT_MAX), 
    ymin(FLT_MAX), ymax(-FLT_MAX),
    zmin(FLT_MAX), zmax(-FLT_MAX) {}
    
bool ScanNetScene::read_ply(const std::filesystem::path& scene_path) {

    // -- helper --
    std::string line;

    // -- get filenames --
    std::string scene_name = scene_path.filename().string();
    std::filesystem::path info_file = scene_path / (scene_name + ".txt");
    std::filesystem::path ply_file = scene_path / (scene_name + "_vh_clean_2.ply");
    std::cout << info_file << std::endl;
    std::cout << ply_file << std::endl;

    // -- read the axis alignment matrix --
    float axis_align[16];
    for (int index = 0; index < 16; index++) {
        int i = index / 4;
        int j = index % 4;
        axis_align[index] = 1.0 * (i==j);
    }
    if (std::filesystem::exists(info_file)){
        std::ifstream info_stream(info_file);
        while (std::getline(info_stream, line)) {
            if (line.rfind("axisAlignment", 0) == 0) { // starts with "axisAlignment"
                std::istringstream iss(line.substr(line.find('=') + 1));
                for (int i = 0; i < 16; i++) {
                    if (!(iss >> axis_align[i])) {
                        std::cerr << "Error parsing axisAlignment\n";
                        return false;
                    }
                }
                break;
            }
        }
        info_stream.close();
    }

    // -- read ply file --
    std::cout << ply_file.string() << std::endl;
    std::ifstream file(ply_file.string(), std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Can not open: " << ply_file << std::endl;
        return false;
    }
    
    // Parse header
    int vertex_count = 0;
    int face_count = 0;
    bool binary_format = false;
    
    while (std::getline(file, line)) {
        std::cout << line << std::endl;
        if (line.find("element vertex") != std::string::npos) {
            vertex_count = std::stoi(line.substr(15));
        } else if (line.find("element face") != std::string::npos) {
            face_count = std::stoi(line.substr(13));
        } else if (line.find("format binary") != std::string::npos) {
            binary_format = true;
        } else if (line == "end_header") {
            break;
        }
    }
    printf("vertex count: %d\n",vertex_count);
    printf("face count: %d\n",face_count);

    // Allocate vectors - now using proper sizing
    pos.resize(vertex_count);
    ftr.resize(vertex_count);
    e0.resize(3*face_count); // triangles
    e1.resize(3*face_count);
    faces.resize(3*face_count);

    // Initialize bounds
    xmin = ymin = zmin = std::numeric_limits<float>::max();
    xmax = ymax = zmax = std::numeric_limits<float>::lowest();
    
    // Read data
    int _ei = 0;
    if (binary_format) {
        for (int i = 0; i < vertex_count; ++i) {
            float x, y, z;
            unsigned char r, g, b, alpha;
            
            // -- read --
            file.read(reinterpret_cast<char*>(&x), sizeof(float));
            file.read(reinterpret_cast<char*>(&y), sizeof(float));
            file.read(reinterpret_cast<char*>(&z), sizeof(float));
            file.read(reinterpret_cast<char*>(&r), sizeof(unsigned char));
            file.read(reinterpret_cast<char*>(&g), sizeof(unsigned char));
            file.read(reinterpret_cast<char*>(&b), sizeof(unsigned char));
            file.read(reinterpret_cast<char*>(&alpha), sizeof(unsigned char));

            // -- axis align --
            float x_new = axis_align[0]*x + axis_align[1]*y + axis_align[2]*z + axis_align[3];
            float y_new = axis_align[4]*x + axis_align[5]*y + axis_align[6]*z + axis_align[7];
            float z_new = axis_align[8]*x + axis_align[9]*y + axis_align[10]*z + axis_align[11];

            // -- update --
            x = x_new;
            y = y_new;
            z = z_new;

            // -- update bounds --
            xmin = std::min(xmin, x);
            xmax = std::max(xmax, x);
            ymin = std::min(ymin, y);
            ymax = std::max(ymax, y);
            zmin = std::min(zmin, z);
            zmax = std::max(zmax, z);
            
            // -- append --
            float3 pos_i = make_float3(x,y,z);
            pos[i] = pos_i;
            float3 ftr_i = make_float3(r/255.0f,g/255.0f,b/255.0f);
            // if (i < 10){
            //     printf("%2.2f %2.2f %2.2f\n",ftr_i.x,ftr_i.y,ftr_i.z);
            // }else{
            //     exit(1);
            // }
            ftr[i] = ftr_i;
        }

        // -- read faces and extract edges --
        for (int i = 0; i < face_count; ++i) {
            unsigned char vertex_count_face;
            file.read(reinterpret_cast<char*>(&vertex_count_face), sizeof(unsigned char));
            if (vertex_count_face != 3){
                printf("vertex count: %d\n",vertex_count_face);
            }
            std::vector<int> vertices(vertex_count_face);
            for (int j = 0; j < vertex_count_face; ++j) {
                file.read(reinterpret_cast<char*>(&vertices[j]), sizeof(int));
                faces[3*i+j] = vertices[j];
            }

            // Extract all edges from this face
            for (int j = 0; j < vertex_count_face; ++j) {
                if (_ei >= (3*face_count)){
                    printf("Broke early! Error! _ei: %d, limit: %d\n", _ei, 3*face_count);
                    break;
                }
                int a = vertices[j];
                int b = vertices[(j + 1) % vertex_count_face];
                e0[_ei] = std::min(a, b);
                e1[_ei] = std::max(a, b);
                _ei += 1;
            }
        }

    } else {
        // ASCII format reading
        for (int i = 0; i < vertex_count; ++i) {
            // -- read --
            float x, y, z;
            int r, g, b, alpha;
            file >> x >> y >> z >> r >> g >> b >> alpha;
            
            // -- axis align --
            float x_new = axis_align[0]*x + axis_align[1]*y + axis_align[2]*z + axis_align[3];
            float y_new = axis_align[4]*x + axis_align[5]*y + axis_align[6]*z + axis_align[7];
            float z_new = axis_align[8]*x + axis_align[9]*y + axis_align[10]*z + axis_align[11];

            // -- update --
            x = x_new;
            y = y_new;
            z = z_new;

            // -- update bounds --
            xmin = std::min(xmin, x);
            xmax = std::max(xmax, x);
            ymin = std::min(ymin, y);
            ymax = std::max(ymax, y);
            zmin = std::min(zmin, z);
            zmax = std::max(zmax, z);

            // -- append --
            // pos[3*i+0] = x;
            // pos[3*i+1] = y;
            // pos[3*i+2] = z;
            // ftr[3*i+0] = r / 255.0f;
            // ftr[3*i+1] = g / 255.0f;
            // ftr[3*i+2] = b / 255.0f;

            // -- append --
            float3 pos_i = make_float3(x,y,z);
            pos[i] = pos_i;
            float3 ftr_i = make_float3(r/255.0f,g/255.0f,b/255.0f);
            ftr[i] = ftr_i;
        }
        
        // Read faces for ASCII format (you'll need to implement this if needed)
        // This was missing in the original ASCII branch
    }
    
    size = vertex_count;
    nfaces = face_count;
    file.close();
    return true;
};

bool ScanNetScene::write_ply_with_fn(const std::filesystem::path& scene_path, 
                            const std::filesystem::path& output_root, PointCloudDataHost& data){
                            // float3* ftrs, float3* pos, uint32_t* edges, 
                            // int nnodes, int nedges, uint8_t* gcolors, uint32_t* labels) {

    // -- helper --
    std::string line;

    // -- make dir if needed --
    std::string scene_name = scene_path.filename().string();
    std::filesystem::path write_path = output_root / scene_name;
    if (!std::filesystem::exists(write_path)) {
        std::filesystem::create_directories(write_path);
    }

    // -- get filenames --
    std::filesystem::path ply_file = write_path / (scene_name + "_vh_clean_2.ply");
    std::cout << ply_file << std::endl;
    //return write_ply(ply_file,ftrs,pos,edges,nnodes,nedges,gcolors,labels);
    return write_ply(ply_file,data);
}


bool ScanNetScene::write_ply(const std::filesystem::path& ply_file, PointCloudDataHost& data) {

    // -- delete existing file if it exists --
    if (std::filesystem::exists(ply_file)) {
        std::filesystem::remove(ply_file);
    }

    // -- open file for writing --
    std::ofstream file(ply_file.string(), std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Can not open: " << ply_file.string() << std::endl;
        return false;
    }

    // -- write PLY header --
    std::string header = "ply\n";
    header += "format binary_little_endian 1.0\n";
    header += "comment MLIB generated\n";
    header += "element vertex " + std::to_string(data.V) + "\n";
    header += "property float x\n";
    header += "property float y\n";
    header += "property float z\n";
    header += "property uchar red\n";
    header += "property uchar green\n";
    header += "property uchar blue\n";
    header += "property uchar alpha\n";
    if (!data.gcolors.empty()){
        header += "property uchar gcolor\n";
    }
    if (!data.labels.empty()){
        header += "property uint label\n";
    }
    header += "element edge " + std::to_string(data.E) + '\n';
    header += "property int vertex1\n";
    header += "property int vertex2\n";
    header += "end_header\n";
    file.write(header.c_str(), header.length());

    // -- write data --
    for (int i = 0; i < data.V; ++i) {
        
        // -- init --
        float x, y, z;
        unsigned char r, g, b, alpha;
        uint32_t label;
        uint8_t gcolor_id;

        // -- unpack --
        float3 ftr_i = data.ftrs[i];
        float3 pos_i = data.pos[i];
        x = pos_i.x;
        y = pos_i.y;
        z = pos_i.z;
        r = static_cast<unsigned char>(ftr_i.x * 255.0f);
        g = static_cast<unsigned char>(ftr_i.y * 255.0f);
        b = static_cast<unsigned char>(ftr_i.z * 255.0f);
        alpha = 255;
        gcolor_id = (!data.gcolors.empty()) ? data.gcolors[i] : 0;
        label =  (!data.labels.empty())? data.labels[i] : 0;
        //printf("label: %ld\n",label);
        // if (i < 10){
        //     printf("x,y,z r,g,b: %2.2f %2.2f %2.2f %d %d %d\n",x,y,z,r,g,b);
        // }

        // -- write --
        file.write(reinterpret_cast<const char*>(&x), sizeof(float));
        file.write(reinterpret_cast<const char*>(&y), sizeof(float));
        file.write(reinterpret_cast<const char*>(&z), sizeof(float));
        file.write(reinterpret_cast<const char*>(&r), sizeof(unsigned char));
        file.write(reinterpret_cast<const char*>(&g), sizeof(unsigned char));
        file.write(reinterpret_cast<const char*>(&b), sizeof(unsigned char));
        file.write(reinterpret_cast<const char*>(&alpha), sizeof(unsigned char));
        // -- write gcolor if provided --
        if (!data.gcolors.empty()) {
            file.write(reinterpret_cast<const char*>(&gcolor_id), sizeof(unsigned char));
        }
        // -- write label if provided --
        if (!data.labels.empty()) {
            file.write(reinterpret_cast<const char*>(&label), sizeof(uint32_t));
        }

    }

    for (int i = 0; i < data.E; ++i){
        //unsigned char len = 2;
        int e0 = data.edges[2*i+0]; // ??
        int e1 = data.edges[2*i+1];
        // if ((i % 10000 == 0) || (i < 10) || (i > (data.E-10))) {        
        //     printf("e0, e1: %d %d\n",e0,e1);
        // }
        //file.write(reinterpret_cast<const char*>(&len), sizeof(unsigned char));
        file.write(reinterpret_cast<const char*>(&e0), sizeof(int));
        file.write(reinterpret_cast<const char*>(&e1), sizeof(int));
    }
    
    file.close();
    return true;

};


// -- write each scene; [nnodes == spix if point-cloud is the superpixel point cloud] --
bool write_spix(const std::vector<std::filesystem::path>& scene_files, 
                const std::filesystem::path& output_root, SuperpixelParams3d& spix_params){
   
    // -- sync before io --
    hipDeviceSynchronize();
    thrust::host_vector<uint32_t> csum_nspix = spix_params.csum_nspix;

    int bx = 0;
    for (const auto& scene_file : scene_files) {

        // -- get batch slice --
        int start_idx = csum_nspix[bx];
        int end_idx = csum_nspix[bx + 1];
        int nspix = end_idx - start_idx;
        thrust::host_vector<float3> mu_app(spix_params.mu_app.begin() + start_idx,
                                        spix_params.mu_app.begin() + end_idx);
        thrust::host_vector<double3> mu_pos(spix_params.mu_pos.begin() + start_idx,
                                            spix_params.mu_pos.begin() + end_idx);
        thrust::host_vector<double3> var_pos(spix_params.var_pos.begin() + start_idx,
                                            spix_params.var_pos.begin() + end_idx);
        thrust::host_vector<double3> cov_pos(spix_params.cov_pos.begin() + start_idx,
                                            spix_params.cov_pos.begin() + end_idx);

        // -- write --
        ScanNetScene scene;
        // if(!scene.write_spix_ply_with_fn(scene_file,output_root,mu_app,mu_pos,var_pos,cov_pos,nspix)){
        //     exit(1);
        // }

        bx += 1;
    }

    return 0;
}


// bool ScanNetScene::write_spix_ply_with_fn(const std::filesystem::path& scene_path, 
//                                   const std::filesystem::path& output_root,
//                                   thrust::host_vector<float3>& ftrs, 
//                                   thrust::host_vector<double3>& pos,
//                                   thrust::host_vector<double3>& var, 
//                                   thrust::host_vector<double3>& cov, int nspix) {

//     // -- helper --
//     std::string line;

//     // -- make dir if needed --
//     std::string scene_name = scene_path.filename().string();
//     std::filesystem::path write_path = output_root / scene_name;
//     if (!std::filesystem::exists(write_path)) {
//         std::filesystem::create_directories(write_path);
//     }
//     // -- get filenames --
//     std::filesystem::path ply_file = write_path / (scene_name + "_spix.ply");
//     std::cout << ply_file << std::endl;
    
//     // thrust::host_vector<uint32_t> border_edges(0); // spoof for now.
//     // thrust::host_vector<uint32_t> border_ptr(0); // spoof for now.
//     return this->write_spix_ply(ply_file,ftrs,pos,var,cov,nspix);

// }


// bool ScanNetScene::write_spix_ply(const std::filesystem::path& ply_file,
//                                   thrust::host_vector<float3>& ftrs, 
//                                   thrust::host_vector<double3>& pos,
//                                   thrust::host_vector<double3>& var, 
//                                   thrust::host_vector<double3>& cov, 
//                                 //   thrust::host_vector<uint32_t>& border_edges,
//                                 //   thrust::host_vector<uint32_t>& border_ptr,
//                                   int nspix) {
    
//     // -- delete existing file if it exists --
//     if (std::filesystem::exists(ply_file)) {
//         std::filesystem::remove(ply_file);
//     }

//     // -- open file for writing --
//     std::ofstream file(ply_file.string(), std::ios::binary);
//     if (!file.is_open()) {
//         std::cerr << "Can not open: " << ply_file.string() << std::endl;
//         return false;
//     }

//     // -- write PLY header --
//     std::string header = "ply\n";
//     header += "format binary_little_endian 1.0\n";
//     header += "comment MLIB generated\n";
//     header += "element vertex " + std::to_string(nspix) + "\n";
//     header += "property float x\n";
//     header += "property float y\n";
//     header += "property float z\n";
//     header += "property float var_x\n";
//     header += "property float var_y\n";
//     header += "property float var_z\n";
//     header += "property float cov_xy\n";
//     header += "property float cov_xz\n";
//     header += "property float cov_yz\n";
//     header += "property uchar red\n";
//     header += "property uchar green\n";
//     header += "property uchar blue\n";
//     header += "property uchar alpha\n";
//     // if (border_edges.size() > 0){
//     //     header += "element edge " + std::to_string(border_edges.size()/2) + '\n';
//     //     header += "property int vertex1\n";
//     //     header += "property int vertex2\n";
//     //     // header += "element face " + std::to_string(border_edges.size()/2) + "\n";
//     //     // header += "property list uchar int vertex_indices\n";
//     // }
//     header += "end_header\n";
//     file.write(header.c_str(), header.length());

//     // -- write data --
//     for (int i = 0; i < nspix; ++i) {
        
//         // -- init --
//         unsigned char r, g, b, alpha;
//         uint32_t label;
//         uint8_t gcolor_id;

//         // -- unpack --
//         float3 _ftrs = ftrs[i];
//         double3 _pos = pos[i];
//         double3 _var = var[i];
//         double3 _cov = cov[i];
//         float x = _pos.x;
//         float y = _pos.y;
//         float z = _pos.z;
//         float var_x = _var.x;
//         float var_y = _var.y;
//         float var_z = _var.z;
//         float cov_x = _cov.x;
//         float cov_y = _cov.y;
//         float cov_z = _cov.z;

//         r = static_cast<unsigned char>(_ftrs.x * 255.0f);
//         g = static_cast<unsigned char>(_ftrs.y * 255.0f);
//         b = static_cast<unsigned char>(_ftrs.z * 255.0f);
//         alpha = 255;
//         //printf("label: %ld\n",label);
//         //printf("x,y,z r,g,b: %2.2f %2.2f %2.2f %2.2f %2.2f %2.2f\n",x,y,z,ftrs[3*i+0],ftrs[3*i+1] ,ftrs[3*i+2] );


//         // -- write --
//         file.write(reinterpret_cast<const char*>(&x), sizeof(float));
//         file.write(reinterpret_cast<const char*>(&y), sizeof(float));
//         file.write(reinterpret_cast<const char*>(&z), sizeof(float));
//         file.write(reinterpret_cast<const char*>(&var_x), sizeof(float));
//         file.write(reinterpret_cast<const char*>(&var_y), sizeof(float));
//         file.write(reinterpret_cast<const char*>(&var_z), sizeof(float));
//         file.write(reinterpret_cast<const char*>(&cov_x), sizeof(float));
//         file.write(reinterpret_cast<const char*>(&cov_y), sizeof(float));
//         file.write(reinterpret_cast<const char*>(&cov_z), sizeof(float));
//         file.write(reinterpret_cast<const char*>(&r), sizeof(unsigned char));
//         file.write(reinterpret_cast<const char*>(&g), sizeof(unsigned char));
//         file.write(reinterpret_cast<const char*>(&b), sizeof(unsigned char));
//         file.write(reinterpret_cast<const char*>(&alpha), sizeof(unsigned char));
//     }


//     file.close();
//     return true;

// };




// PointCloudData read_scene(const std::vector<std::filesystem::path>& scene_files){

//     // -- read sizes --
//     int batchsize = scene_files.size();

//     // -- batch offset --
//     std::vector<int> ptr_host(batchsize+1,0);
//     std::vector<int> eptr_host(batchsize+1,0);
//     std::vector<int> face_ptr_host(batchsize+1,0);
    
//     // First pass: get sizes
//     int _ix = 1;
//     for (const auto& scene_file : scene_files) {
//         ptr_host[_ix] = get_vertex_count(scene_file)+ptr_host[_ix-1];
//         face_ptr_host[_ix] = get_face_count(scene_file)+face_ptr_host[_ix-1];
//         _ix++;
//     }
//     int total_nodes = ptr_host[batchsize];
//     int total_faces = face_ptr_host[batchsize];
//     printf("total nodes, total faces: %d, %d\n",total_nodes,total_faces);

//     // Reserve space for all points (host vectors)
//     std::vector<float3> ftrs_host(total_nodes);
//     std::vector<float3> pos_host(total_nodes);
//     std::vector<float> dim_sizes_host(6*batchsize,0);
//     std::vector<uint8_t> bids_host(total_nodes,0);
//     std::vector<uint32_t> faces_host(3*total_faces,0);
//     thrust::device_vector<uint32_t> edges{};
//     std::vector<uint8_t> ebids_host{};

//     // Second pass: append data
//     int _bx = 0;
//     _ix = 0;
//     for (const auto& scene_file : scene_files) {

//         // -- .. --
//         ScanNetScene scene;
//         if(!scene.read_ply(scene_file)){
//             exit(1);
//         }

//         // -- .. --
//         memcpy(&ftrs[3*_ix], scene.ftr.data(), 3 * scene.size * sizeof(float));
//         memcpy(&pos[3*_ix], scene.pos.data(), 3 * scene.size * sizeof(float));
//         std::fill(bids.begin() + _ix, bids.begin() + _ix + scene.size, _bx);

//         int nfaces_batch = face_ptr_cpu[_bx+1] - face_ptr_cpu[_bx];
//         memcpy(&faces[3*face_ptr_cpu[_bx]], scene.faces.data(), 3 * nfaces_batch * sizeof(uint32_t));

//         // -- extract edges from edge-pairs --
//         thrust::device_vector<uint32_t> edges_b = extract_edges_from_pairs(scene.e0,scene.e1);
//         size_t _size = edges.size();
//         // edges.resize(_size + edges_b.size());
//         // hipDeviceSynchronize();
//         // thrust::copy(edges_b.begin(), edges_b.end(), edges.begin() + _size);
//         edges.insert(edges.end(), edges_b.begin(), edges_b.end());
//         //ebids.resize(_size/2 + edges_b.size()/2, _bx);
//         ebids.insert(ebids.end(), edges_b.size()/2, _bx);
//         eptr_cpu[_bx+1] = eptr_cpu[_bx]+edges_b.size()/2;
//         printf("eptr_cpu: %d %d\n", eptr_cpu[_bx],eptr_cpu[_bx+1]);

//         // -- .. --
//         dim_sizes[6*_bx+0] = scene.xmin;
//         dim_sizes[6*_bx+1] = scene.xmax;
//         dim_sizes[6*_bx+2] = scene.ymin;
//         dim_sizes[6*_bx+3] = scene.ymax;
//         dim_sizes[6*_bx+4] = scene.zmin;
//         dim_sizes[6*_bx+5] = scene.zmax;
//         _ix += scene.size;
//         _bx += 1;

//         // // -- .. --
//         // ScanNetScene scene;
//         // if(!scene.read_ply(scene_file)){
//         //     exit(1);
//         // }

//         // // -- copy features and positions --
//         // for(int i = 0; i < scene.size; i++) {
//         //     ftrs_host[_ix + i] = make_float3(scene.ftr[3*i], scene.ftr[3*i+1], scene.ftr[3*i+2]);
//         //     pos_host[_ix + i] = make_float3(scene.pos[3*i], scene.pos[3*i+1], scene.pos[3*i+2]);
//         // }
        
//         // // -- set batch ids --
//         // std::fill(bids_host.begin() + _ix, bids_host.begin() + _ix + scene.size, _bx);

//         // // -- copy faces --
//         // int nfaces_batch = face_ptr_host[_bx+1] - face_ptr_host[_bx];
//         // memcpy(&faces_host[3*face_ptr_host[_bx]], scene.faces.data(), 3 * nfaces_batch * sizeof(uint32_t));

//         // // -- extract edges from edge-pairs --
//         // thrust::device_vector<uint32_t> edges_b = extract_edges_from_pairs(scene.e0,scene.e1);
//         // edges.insert(edges.end(), edges_b.begin(), edges_b.end());
//         // ebids_host.insert(ebids_host.end(), edges_b.size()/2, _bx);
//         // eptr_host[_bx+1] = eptr_host[_bx]+edges_b.size()/2;
//         // printf("eptr_host: %d %d\n", eptr_host[_bx],eptr_host[_bx+1]);

//         // // -- copy dimension sizes --
//         // dim_sizes_host[6*_bx+0] = scene.xmin;
//         // dim_sizes_host[6*_bx+1] = scene.xmax;
//         // dim_sizes_host[6*_bx+2] = scene.ymin;
//         // dim_sizes_host[6*_bx+3] = scene.ymax;
//         // dim_sizes_host[6*_bx+4] = scene.zmin;
//         // dim_sizes_host[6*_bx+5] = scene.zmax;
//         // _ix += scene.size;
//         // _bx += 1;
//     }

//     // -- view --
//     printf("_ix: %d\n",_ix);
//     std::cout << "=== Scene Batch Info ===" << std::endl;
//     std::cout << "Batch size: " << batchsize << std::endl;
//     std::cout << "Total nodes: " << total_nodes << std::endl;

//     // Print per-scene info
//     for (int i = 0; i < batchsize; ++i) {
//         std::cout << "Scene " << i << ": " << (ptr_host[i+1] - ptr_host[i]) << " points" << std::endl;
//         std::cout << "  Bounds: [" << dim_sizes_host[6*i+0] << ", " << dim_sizes_host[6*i+1] << "] "
//                 << "[" << dim_sizes_host[6*i+2] << ", " << dim_sizes_host[6*i+3] << "] "
//                 << "[" << dim_sizes_host[6*i+4] << ", " << dim_sizes_host[6*i+5] << "]" << std::endl;
//     }

//     // Print first few points
//     std::cout << "First 3 points:" << std::endl;
//     for (int i = 0; i < std::min(3, total_nodes); i++) {
//         std::cout << "  Pos: (" << pos_host[i].x << ", " << pos_host[i].y << ", " << pos_host[i].z << ")" << std::endl;
//         std::cout << "  RGB: (" << ftrs_host[i].x << ", " << ftrs_host[i].y << ", " << ftrs_host[i].z << ")" << std::endl;
//     }

//     // Create device vectors and copy data
//     int nedges = eptr_host[batchsize];
//     printf("nedges: %d,%d\n",nedges,ebids_host.size());
    
//     thrust::device_vector<uint8_t> gcolors_host;
//     thrust::device_vector<uint32_t> csr_edges_host;
//     thrust::device_vector<uint32_t> csr_eptr_host;


//     PointCloudData result(
//         ftrs_host, pos_host, faces_host, edges, 
//         bids_host, ptr_host, eptr_host, fptr_host,
//         dim_sizes_host, 0, batchsize, total_nodes, nedges, total_faces
//     );

//     return result;

// }



    // Create device vectors by moving/copying from host data
    // thrust::device_vector<float3> ftrs_device(ftrs_host.begin(), ftrs_host.end());
    // thrust::device_vector<float3> pos_device(pos_host.begin(), pos_host.end());
    // thrust::device_vector<uint32_t> faces_device(faces_host.begin(), faces_host.end());
    // thrust::device_vector<uint8_t> gcolors_device; // Empty for now - you might want to populate this
    // thrust::device_vector<uint32_t> csr_edges_device(std::move(edges)); // Move the edges we built
    // thrust::device_vector<uint32_t> csr_eptr_device; // You might want to populate this based on your CSR needs
    // thrust::device_vector<uint8_t> bids_device(bids_host.begin(), bids_host.end());
    // thrust::device_vector<int> ptr_device(ptr_host, ptr_host + batchsize + 1);
    // thrust::device_vector<int> eptr_device(eptr_host, eptr_host + batchsize + 1);
    // thrust::device_vector<int> fptr_device(face_ptr_host, face_ptr_host + batchsize + 1);
    // // thrust::device_vector<float> dim_sizes_device(dim_sizes_host.begin(), dim_sizes_host.end());

    // // Calculate edge count for the struct
    // int E = csr_edges_device.size() / 2; // assuming edges are stored as pairs
    
    // // // Create and return PointCloudData struct
    // return PointCloudData(
    //     std::move(ftrs_device),
    //     std::move(pos_device), 
    //     std::move(faces_device),
    //     std::move(gcolors_device),
    //     std::move(csr_edges_device),
    //     std::move(csr_eptr_device),
    //     std::move(bids_device),
    //     std::move(ptr_device),
    //     std::move(eptr_device),
    //     std::move(fptr_device),
    //     std::move(dim_sizes_device),
    //     0, // gchrome - set to appropriate value
    //     batchsize, // B
    //     total_nodes, // V
    //     E, // E
    //     total_faces // F
    // );


// -- read each scene --
// std::tuple<float3*,float3*,uint32_t*,uint8_t*,uint8_t*,int*,int*,float*,uint32_t*,int*>
// read_scene(const std::vector<std::filesystem::path>& scene_files){

//     // -- read sizes --
//     int batchsize = scene_files.size();
//     int* ptr_cpu = (int*)malloc((batchsize+1) * sizeof(int));
//     int* face_ptr_cpu = (int*)malloc((batchsize+1) * sizeof(int));
    
//     // First pass: get sizes
//     int _ix = 1;
//     ptr_cpu[0] = 0;
//     face_ptr_cpu[0] = 0;
//     for (const auto& scene_file : scene_files) {
//         ptr_cpu[_ix] = get_vertex_count(scene_file)+ptr_cpu[_ix-1];
//         face_ptr_cpu[_ix] = get_face_count(scene_file)+face_ptr_cpu[_ix-1];
//         _ix++;
//     }
//     int total_nodes = ptr_cpu[batchsize];
//     int total_faces = face_ptr_cpu[batchsize];
//     printf("total nodes, total faces: %d, %d\n",total_nodes,total_faces);

//     // Reserve space for all points
//     std::vector<float> ftrs(3*total_nodes,0);
//     std::vector<float> pos(3*total_nodes,0);
//     std::vector<float> dim_sizes(6*batchsize,0);
//     std::vector<uint8_t> bids(total_nodes,0);
//     std::vector<uint32_t> faces(3*total_faces,0);
//     thrust::device_vector<uint32_t> edges{};
//     std::vector<uint8_t> ebids{};
//     int* eptr_cpu = (int*)malloc((batchsize+1) * sizeof(int));
//     eptr_cpu[0] = 0;

//     // Second pass: append data
//     int _bx = 0;
//     _ix = 0;
//     for (const auto& scene_file : scene_files) {

//         // -- .. --
//         ScanNetScene scene;
//         if(!scene.read_ply(scene_file)){
//             exit(1);
//         }

//         // -- .. --
//         memcpy(&ftrs[3*_ix], scene.ftr.data(), 3 * scene.size * sizeof(float));
//         memcpy(&pos[3*_ix], scene.pos.data(), 3 * scene.size * sizeof(float));
//         std::fill(bids.begin() + _ix, bids.begin() + _ix + scene.size, _bx);

//         int nfaces_batch = face_ptr_cpu[_bx+1] - face_ptr_cpu[_bx];
//         memcpy(&faces[3*face_ptr_cpu[_bx]], scene.faces.data(), 3 * nfaces_batch * sizeof(uint32_t));

//         // -- extract edges from edge-pairs --
//         thrust::device_vector<uint32_t> edges_b = extract_edges_from_pairs(scene.e0,scene.e1);
//         size_t _size = edges.size();
//         // edges.resize(_size + edges_b.size());
//         // hipDeviceSynchronize();
//         // thrust::copy(edges_b.begin(), edges_b.end(), edges.begin() + _size);
//         edges.insert(edges.end(), edges_b.begin(), edges_b.end());
//         //ebids.resize(_size/2 + edges_b.size()/2, _bx);
//         ebids.insert(ebids.end(), edges_b.size()/2, _bx);
//         eptr_cpu[_bx+1] = eptr_cpu[_bx]+edges_b.size()/2;
//         printf("eptr_cpu: %d %d\n", eptr_cpu[_bx],eptr_cpu[_bx+1]);

//         // -- .. --
//         dim_sizes[6*_bx+0] = scene.xmin;
//         dim_sizes[6*_bx+1] = scene.xmax;
//         dim_sizes[6*_bx+2] = scene.ymin;
//         dim_sizes[6*_bx+3] = scene.ymax;
//         dim_sizes[6*_bx+4] = scene.zmin;
//         dim_sizes[6*_bx+5] = scene.zmax;
//         _ix += scene.size;
//         _bx += 1;
//     }

//     // -- view --
//     printf("_ix: %d\n",_ix);
//     std::cout << "=== Scene Batch Info ===" << std::endl;
//     std::cout << "Batch size: " << batchsize << std::endl;
//     std::cout << "Total nodes: " << total_nodes << std::endl;

//     // Print per-scene info
//     int offset = 0;
//     for (int i = 0; i < batchsize; ++i) {
//         std::cout << "Scene " << i << ": " << (ptr_cpu[i+1] - ptr_cpu[i]) << " points" << std::endl;
//         std::cout << "  Bounds: [" << dim_sizes[6*i+0] << ", " << dim_sizes[6*i+1] << "] "
//                 << "[" << dim_sizes[6*i+2] << ", " << dim_sizes[6*i+3] << "] "
//                 << "[" << dim_sizes[6*i+4] << ", " << dim_sizes[6*i+5] << "]" << std::endl;
//     }

//     // Print first few points
//     std::cout << "First 3 points:" << std::endl;
//     for (int i = 0; i < std::min(9, (int)pos.size()); i += 3) {
//         std::cout << "  Pos: (" << pos[i] << ", " << pos[i+1] << ", " << pos[i+2] << ")" << std::endl;
//         std::cout << "  RGB: (" << ftrs[i] << ", " << ftrs[i+1] << ", " << ftrs[i+2] << ")" << std::endl;
//     }

//     // -- copy --
//     int nedges = eptr_cpu[batchsize];
//     printf("nedges: %d,%d\n",nedges,ebids.size());
//     float3* ftrs_cu = (float3*)easy_allocate(total_nodes,sizeof(float3));
//     float3* pos_cu = (float3*)easy_allocate(total_nodes,sizeof(float3));
//     uint32_t* edges_cu = (uint32_t*)easy_allocate(2*nedges,sizeof(uint32_t));
//     uint8_t* bids_cu = (uint8_t*)easy_allocate(total_nodes,sizeof(uint8_t));
//     uint8_t* ebids_cu = (uint8_t*)easy_allocate(nedges,sizeof(uint8_t));
//     int* ptr_cu = (int*)easy_allocate(batchsize+1,sizeof(int));
//     int* eptr_cu = (int*)easy_allocate(batchsize+1,sizeof(int));
//     float* dim_sizes_cu = (float*)easy_allocate(6*batchsize,sizeof(float));

//     uint32_t* faces_cu = (uint32_t*)easy_allocate(total_faces,sizeof(uint32_t));
//     int* face_ptr_cu = (int*)easy_allocate(batchsize+1,sizeof(int));


//     hipDeviceSynchronize();
//     hipMemcpy(ftrs_cu,thrust::raw_pointer_cast(ftrs.data()),total_nodes*sizeof(float3),hipMemcpyHostToDevice);
//     hipMemcpy(pos_cu,thrust::raw_pointer_cast(pos.data()),total_nodes*sizeof(float3),hipMemcpyHostToDevice);
//     hipMemcpy(edges_cu,thrust::raw_pointer_cast(edges.data()),2*nedges*sizeof(uint32_t),hipMemcpyDeviceToDevice);
//     hipMemcpy(bids_cu,thrust::raw_pointer_cast(bids.data()),total_nodes*sizeof(uint8_t),hipMemcpyHostToDevice);
//     hipMemcpy(ebids_cu,thrust::raw_pointer_cast(ebids.data()),nedges*sizeof(uint8_t),hipMemcpyHostToDevice);
//     hipMemcpy(ptr_cu,ptr_cpu,(batchsize+1)*sizeof(int),hipMemcpyHostToDevice);
//     hipMemcpy(eptr_cu,eptr_cpu,(batchsize+1)*sizeof(int),hipMemcpyHostToDevice);
//     hipMemcpy(dim_sizes_cu,thrust::raw_pointer_cast(dim_sizes.data()),6*batchsize*sizeof(float),hipMemcpyHostToDevice);

//     hipMemcpy(faces_cu,thrust::raw_pointer_cast(faces.data()),total_faces*sizeof(uint32_t),hipMemcpyHostToDevice);
//     hipMemcpy(face_ptr_cu,face_ptr_cpu,(batchsize+1)*sizeof(int),hipMemcpyHostToDevice);

//     // -- free --
//     free(ptr_cpu);
//     free(eptr_cpu);

//     return std::tuple(ftrs_cu,pos_cu,edges_cu,bids_cu,ebids_cu,ptr_cu,eptr_cu,dim_sizes_cu,faces_cu,face_ptr_cu);

// }


// if (border_edges.size()>0){
//     // uint32_t nedges = border_edges.size()/2;
//     // for (uint32_t i = 0; i < nedges; ++i) {
//     //     uint32_t start = border_ptr[i];
//     //     uint32_t end = border_ptr[i+1];
//     //     unsigned char spix_size = end - start;
//     //     file.write(reinterpret_cast<char*>(&spix_size), sizeof(unsigned char));
//     //     for (uint32_t index=start; index < end; index++){
//     //         uint32_t vertex_id = border_edges[index];
//     //         file.write(reinterpret_cast<char*>(&vertex_id), sizeof(int));
//     //         printf("[%d,%d] %d\n",index,spix_size,vertex_id);
//     //     }
//     // }
//     uint32_t nedges = border_edges.size()/2;
//     for (uint32_t i = 0; i < nedges; ++i){
//         uint32_t e0 = border_edges[2*i+0]; // ??
//         uint32_t e1 = border_edges[2*i+1];
//         if ((i % 10000 == 0) || (i < 10) || (i > (nedges-10))) {        
//             printf("e0, e1: %d %d\n",e0,e1);
//         }
//         //file.write(reinterpret_cast<const char*>(&len), sizeof(unsigned char));
//         file.write(reinterpret_cast<const char*>(&e0), sizeof(int));
//         file.write(reinterpret_cast<const char*>(&e1), sizeof(int));
//     }
    
// }


// bool write_ply_csr_edges(const std::filesystem::path& ply_file,
//                         float* ftrs, float* pos, uint32_t* csr_edges,  uint32_t* csr_eptr,  
//                         int V, int E){

//     // -- delete existing file if it exists --
//     if (std::filesystem::exists(ply_file)) {
//         std::filesystem::remove(ply_file);
//     }
//     bool* gcolors = nullptr;
//     uint32_t* labels = nullptr;

//     // -- open file for writing --
//     std::ofstream file(ply_file.string(), std::ios::binary);
//     if (!file.is_open()) {
//         std::cerr << "Can not open: " << ply_file.string() << std::endl;
//         return false;
//     }

//     // -- write PLY header --
//     std::string header = "ply\n";
//     header += "format binary_little_endian 1.0\n";
//     header += "comment MLIB generated\n";
//     header += "element vertex " + std::to_string(V) + "\n";
//     header += "property float x\n";
//     header += "property float y\n";
//     header += "property float z\n";
//     header += "property uchar red\n";
//     header += "property uchar green\n";
//     header += "property uchar blue\n";
//     header += "property uchar alpha\n";
//     if (gcolors == nullptr){
//         header += "property uchar gcolor\n";
//     }
//     if (labels != nullptr) {
//         header += "property uint label\n";
//     }
//     header += "element edge " + std::to_string(E) + '\n';
//     header += "property int vertex1\n";
//     header += "property int vertex2\n";
//     header += "end_header\n";
//     file.write(header.c_str(), header.length());

//     // -- write data --
//     for (int i = 0; i < V; ++i) {
        
//         // -- init --
//         float x, y, z;
//         unsigned char r, g, b, alpha;
//         uint32_t label;
//         uint8_t gcolor_id;

//         // -- unpack --
//         x = pos[3*i+0];
//         y = pos[3*i+1];
//         z = pos[3*i+2];
//         r = static_cast<unsigned char>(ftrs[3*i+0] * 255.0f);
//         g = static_cast<unsigned char>(ftrs[3*i+1] * 255.0f);
//         b = static_cast<unsigned char>(ftrs[3*i+2] * 255.0f);
//         alpha = 255;
//         gcolor_id = (gcolors != nullptr) ? gcolors[i] : 0;
//         label = (labels != nullptr) ? static_cast<uint32_t>(labels[i]) : 0;
//         //printf("label: %ld\n",label);
//         //printf("x,y,z r,g,b: %2.2f %2.2f %2.2f %2.2f %2.2f %2.2f\n",x,y,z,ftrs[3*i+0],ftrs[3*i+1] ,ftrs[3*i+2] );


//         // -- write --
//         file.write(reinterpret_cast<const char*>(&x), sizeof(float));
//         file.write(reinterpret_cast<const char*>(&y), sizeof(float));
//         file.write(reinterpret_cast<const char*>(&z), sizeof(float));
//         file.write(reinterpret_cast<const char*>(&r), sizeof(unsigned char));
//         file.write(reinterpret_cast<const char*>(&g), sizeof(unsigned char));
//         file.write(reinterpret_cast<const char*>(&b), sizeof(unsigned char));
//         file.write(reinterpret_cast<const char*>(&alpha), sizeof(unsigned char));
//         // -- write gcolor if provided --
//         if (gcolors != nullptr) {
//             file.write(reinterpret_cast<const char*>(&gcolor_id), sizeof(unsigned char));
//         }
//         // -- write label if provided --
//         if (labels != nullptr) {
//             file.write(reinterpret_cast<const char*>(&label), sizeof(uint32_t));
//         }

//     }

//     for (int i = 0; i < E; ++i){
//         //unsigned char len = 2;
//         int e0 = edges[2*i+0]; // ??
//         int e1 = edges[2*i+1];

//         for (int jx = start; jx < end; jx++){
//             file.write(reinterpret_cast<const char*>(&e0), sizeof(int));
//             file.write(reinterpret_cast<const char*>(&e1), sizeof(int));
//         }
//     }
    
//     file.close();
//     return true;                   

// }



// bool ScanNetScene::read_ply(const std::filesystem::path& scene_path) {

//     // -- helper --
//     std::string line;

//     // -- get filenames --
//     std::string scene_name = scene_path.filename().string();
//     std::filesystem::path info_file = scene_path / (scene_name + ".txt");
//     std::filesystem::path ply_file = scene_path / (scene_name + "_vh_clean_2.ply");
//     std::cout << info_file << std::endl;
//     std::cout << ply_file << std::endl;

//     // -- read the axis alignment matrix --
//     float axis_align[16];
//     for (int index = 0; index < 16; index++) {
//         int i = index / 4;
//         int j = index % 4;
//         axis_align[index] = 1.0 * (i==j);
//     }
//     if (std::filesystem::exists(info_file)){
//         std::ifstream info_stream(info_file);
//         while (std::getline(info_stream, line)) {
//             if (line.rfind("axisAlignment", 0) == 0) { // starts with "axisAlignment"
//                 std::istringstream iss(line.substr(line.find('=') + 1));
//                 for (int i = 0; i < 16; i++) {
//                     if (!(iss >> axis_align[i])) {
//                         std::cerr << "Error parsing axisAlignment\n";
//                         return 1;
//                     }
//                 }
//                 break;
//             }
//         }
//         info_stream.close();
//     }

//     // // Print result to verify
//     // for (int i = 0; i < 16; i++) {
//     //     std::cout << axis_align[i] << (i % 4 == 3 ? "\n" : " ");
//     // }

//     // -- read ply file --
//     std::cout << ply_file.string() << std::endl;
//     std::ifstream file(ply_file.string(), std::ios::binary);
//     if (!file.is_open()) {
//         std::cerr << "Can not open: " << ply_file << std::endl;
//         return false;
//     }
    
//     // Parse header
//     int vertex_count = 0;
//     int face_count = 0;
//     bool binary_format = false;
    
//     while (std::getline(file, line)) {
//         std::cout << line << std::endl;
//         if (line.find("element vertex") != std::string::npos) {
//             vertex_count = std::stoi(line.substr(15));
//         } else if (line.find("element face") != std::string::npos) {
//             face_count = std::stoi(line.substr(13));
//         } else if (line.find("format binary") != std::string::npos) {
//             binary_format = true;
//         } else if (line == "end_header") {
//             break;
//         }
//     }
//     printf("vertex count: %d\n",vertex_count);
//     printf("face count: %d\n",face_count);


//     // Allocate vectors
//     pos.reserve(3*vertex_count);
//     ftr.reserve(3*vertex_count);
//     e0.resize(3*face_count); // triangles
//     e1.resize(3*face_count);
//     faces.resize(3*face_count);

    
//     // Read data
//     int _ei = 0;
//     if (binary_format) {
//         for (int i = 0; i < vertex_count; ++i) {
//             float x, y, z;
//             unsigned char r, g, b, alpha;
            
//             // -- read --
//             file.read(reinterpret_cast<char*>(&x), sizeof(float));
//             file.read(reinterpret_cast<char*>(&y), sizeof(float));
//             file.read(reinterpret_cast<char*>(&z), sizeof(float));
//             file.read(reinterpret_cast<char*>(&r), sizeof(unsigned char));
//             file.read(reinterpret_cast<char*>(&g), sizeof(unsigned char));
//             file.read(reinterpret_cast<char*>(&b), sizeof(unsigned char));
//             file.read(reinterpret_cast<char*>(&alpha), sizeof(unsigned char));

//             // -- axis align --
//             float x_new = axis_align[0]*x + axis_align[1]*y + axis_align[2]*z + axis_align[3];
//             float y_new = axis_align[4]*x + axis_align[5]*y + axis_align[6]*z + axis_align[7];
//             float z_new = axis_align[8]*x + axis_align[9]*y + axis_align[10]*z + axis_align[11];

//             // -- update --
//             x = x_new;
//             y = y_new;
//             z = z_new;

//             // -- update bounds --
//             xmin = std::min(xmin, x);
//             xmax = std::max(xmax, x);
//             ymin = std::min(ymin, y);
//             ymax = std::max(ymax, y);
//             zmin = std::min(zmin, z);
//             zmax = std::max(zmax, z);
            
//             // -- append --
//             pos[3*i+0] = x;
//             pos[3*i+1] = y;
//             pos[3*i+2] = z;
//             ftr[3*i+0] = r / 255.0f;
//             ftr[3*i+1] = g / 255.0f;
//             ftr[3*i+2] = b / 255.0f;
//             // e0[_ei] = i;
//             // e1[_ei] = i;
//             // _ei++;
//         }

//         // -- read edges --
//         for (int i = 0; i < face_count; ++i) {
//             unsigned char vertex_count;
//             file.read(reinterpret_cast<char*>(&vertex_count), sizeof(unsigned char));
//             if (vertex_count != 3){
//                 printf("vertex count: %d\n",vertex_count);
//             }
//             std::vector<int> vertices(vertex_count);
//             for (int j = 0; j < vertex_count; ++j) {
//                 file.read(reinterpret_cast<char*>(&vertices[j]), sizeof(int));
//                 faces[3*i+j] = vertices[j];
//             }

//             // Extract all edges from this face
//             for (int j = 0; j < vertex_count; ++j) {
//                 if ((_ei-vertex_count) >= (6*face_count)){
//                     printf("Broke early! Error!\n");
//                     exit(1);
//                 }
//                 int a = vertices[j];
//                 int b = vertices[(j + 1) % vertex_count];
//                 // if (a == b){ 
//                 //     printf("self loop? %d\n",a);
//                 //     continue; 
//                 // }
//                 e0[_ei] = std::min(a, b);
//                 e1[_ei] = std::max(a, b);
//                 _ei += 1;
//             }
//         }
//         //printf("n pairs: %d\n",_ei);
//         assert(_ei == e0.size()); // must fill the vector.

//     } else {
//         for (int i = 0; i < vertex_count; ++i) {
//             // -- read --
//             float x, y, z;
//             int r, g, b, alpha;
//             file >> x >> y >> z >> r >> g >> b >> alpha;
//             //printf("x,y,z: %.2f %.2f %.2f\n",x,y,z);
            
//             // -- axis align --
//             float x_new = axis_align[0]*x + axis_align[1]*y + axis_align[2]*z + axis_align[3];
//             float y_new = axis_align[4]*x + axis_align[5]*y + axis_align[6]*z + axis_align[7];
//             float z_new = axis_align[8]*x + axis_align[9]*y + axis_align[10]*z + axis_align[11];

//             // -- update --
//             x = x_new;
//             y = y_new;
//             z = z_new;

//             // -- update bounds --
//             xmin = std::min(xmin, x);
//             xmax = std::max(xmax, x);
//             ymin = std::min(ymin, y);
//             ymax = std::max(ymax, y);
//             zmin = std::min(zmin, z);
//             zmax = std::max(zmax, z);

//             // -- append --
//             pos[3*i+0] = x;
//             pos[3*i+1] = y;
//             pos[3*i+2] = z;
//             ftr[3*i+0] = r / 255.0f;
//             ftr[3*i+1] = g / 255.0f;
//             ftr[3*i+2] = b / 255.0f;
//         }
//     }
    
//     size = vertex_count;
//     nfaces = face_count;
//     //nfaces = 3*face_count+vertex_count;
//     file.close();
//     return true;
// };
