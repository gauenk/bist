#include "hip/hip_runtime.h"
/*************************************************************************************************
 * 
 * 
 *    Articulation Points are another name for "NOT" Simple Ponts from the 2D Superpixel Papers 
 *    Check-out: "A Fast Method for Inferring High-Quality Simply-Connected Superpixels"
 *    
 *    In 3D, finding these points is a fundamentally challenging problem. If we have a cluster of
 *    vertices and we are checking if vertex v is simple, we need to check 
 *    if there is ANY path to connect ALL the remaining nodes. This is a global(-ish) condition 
 *    and we only get to check it locally. In 2D, the problem is simple since the dimension is 
 *    only 2D and we know the connectivity between our neighbors.
 * 
 *    We do a 2-hop approximation. For point-clouds lifted from a triangular mesh (ScanNetv2),
 *    I think it's "good enough" for this method..
 * 
 * 
 **************************************************************************************************/


#include "articulation_points.h"

__device__ bool check_2hop_connectivity(
    uint32_t n1, uint32_t n2, uint32_t avoid_vertex,
    const uint32_t* csr_edges, const uint32_t* csr_ptr,
    const uint32_t* neighbors_cache, int num_neighbors_cache
) {
    // 1-hop: Check if n1 and n2 are directly connected
    // Search in the avoid_vertex's neighbor list (cached in shared memory)
    bool n1_found = false, n2_found = false;
    for (int i = 0; i < num_neighbors_cache; i++) {
        if (neighbors_cache[i] == n1) n1_found = true;
        if (neighbors_cache[i] == n2) n2_found = true;
    }
    
    // If both are neighbors of avoid_vertex, check if they're direct neighbors
    if (n1_found && n2_found) {
        // Check global CSR for direct n1-n2 connection
        int start_n1 = csr_ptr[n1];
        int end_n1 = csr_ptr[n1 + 1];
        for (int i = start_n1; i < end_n1; i++) {
            if (csr_edges[i] == n2) return true;
        }
    }
    
    // 2-hop: Check if n1 and n2 share any common neighbors (excluding avoid_vertex)
    int start_n1 = csr_ptr[n1];
    int end_n1 = csr_ptr[n1 + 1];
    
    for (int i = start_n1; i < end_n1; i++) {
        uint32_t common = csr_edges[i];
        if (common == avoid_vertex) continue;
        
        // Check if 'common' is also neighbor of n2
        int start_n2 = csr_ptr[n2];
        int end_n2 = csr_ptr[n2 + 1];
        
        for (int j = start_n2; j < end_n2; j++) {
            if (csr_edges[j] == common) {
                return true;  // Found 2-hop path: n1-common-n2
            }
        }
    }
    
    return false;  // No 2-hop connection found
}

// i -> (row, col), strict upper triangle (excluding diagonal)
__device__ void upper_strict_from_index(uint32_t n, int i, int& row, int& col) {
    int n2 = n - 1;                    // map to an (n-1)x(n-1) upper-inc problem
    int N2 = n2*(n2+1)/2;              // equals n*(n-1)/2
    int j  = N2 - 1 - i;
    int rprime = ( (int)std::floor((std::sqrt(8.0*j+1)-1)/2) );
    int cprime = j - rprime*(rprime+1)/2;
    row = (n2 - 1) - rprime;                 // same row
    col = (n2 - 1) - cprime + 1;             // shift right by 1 to skip the diagonal
}




// __global__ void approximate_articulation_points(
//     const uint32_t* labels,  // Cluster Labels
//     const bool*     border,
//     const float3* pos,
//     const uint32_t* csr_edges,           // 1-hop neighbor data
//     const uint32_t* csr_ptr,             // CSR pointers
//     bool* is_simple_point,                // Output: true if simple point
//     uint8_t* num_neq, // Output: Num p
//     uint8_t* gcolors,
//     uint8_t gchrome,
//     uint32_t V                   // Number of vertices
// ) {
    

//     // Warp and thread identification
//     uint32_t vertex = (blockIdx.x * blockDim.x + threadIdx.x);
//     if (vertex >= V) return;

//     uint32_t my_label = labels[vertex];
//     uint8_t gcolor = gcolors[vertex];
//     if (gcolor != gchrome){ return; }
//     if (!border[vertex]) { return; }

//     uint32_t start = csr_ptr[vertex];
//     uint32_t end = csr_ptr[vertex+1];
//     // if ((end - start) > 3){
//     //     printf("vertex[%d]: # of edges %d %d\n",vertex,start,end);
//     // }
//     assert( (end - start) <= 3);
//     //bool any_neq = false;
//     uint8_t num_eq = 0;
//     uint8_t num_neq_v = 0;
//     uint32_t eq_neigh[2];
//     for(int index = start; index < end; index++){
//         uint32_t neigh = csr_edges[index];
//         uint32_t neigh_vertex = labels[neigh];
//         bool neq = neigh_vertex != my_label;
//         num_neq_v += neq;
//         if (!neq){
//             if(num_eq < 2){
//                 eq_neigh[num_eq] = neigh;
//             }
//             num_eq = num_eq + 1;
//         }
//     }
    
//     // -- .. --
//     assert(num_eq <= 2);
//     if (num_eq == 1){
//         is_simple_point[vertex] = 0;
//         return;
//     }

//     // -- .. --
//     bool all_eq = true;
//     for(int neigh_ix=0; neigh_ix < 2; neigh_ix++){
//         uint32_t neigh = eq_neigh[neigh_ix];
//         uint32_t start = csr_ptr[neigh];
//         uint32_t end = csr_ptr[neigh+1];
//         bool found_neq = false;
//         for(int index = start; index < end; index++){
//             uint32_t nneigh = csr_edges[index];
//             uint32_t nneigh_vertex = labels[nneigh];
//             bool neq = nneigh_vertex != my_label;
//             found_neq = found_neq || neq;
//         }
//         all_eq = all_eq && (!found_neq);
//     }


//     is_simple_point[vertex] = 1;
// }


__device__
float get_angle(float3 origin, float3 pos0, float3 pos1){
    float3 v0 = make_float3(pos0.x - origin.x, pos0.y - origin.y, pos0.z - origin.z);
    float3 v1 = make_float3(pos1.x - origin.x, pos1.y - origin.y, pos1.z - origin.z);
    float dot = v0.x*v1.x + v0.y*v1.y + v0.z*v1.z;
    float len0 = sqrtf(v0.x*v0.x + v0.y*v0.y + v0.z*v0.z);
    float len1 = sqrtf(v1.x*v1.x + v1.y*v1.y + v1.z*v1.z);
    float cos_angle = dot / (len0 * len1 + 1e-6);
    cos_angle = fminf(fmaxf(cos_angle,-1.0f),1.0f);
    float angle = acosf(cos_angle);
    return angle;
}

__device__
float get_signed_angle(float3 origin, float3 from, float3 to, float3 face_normal) {
    // Vectors from origin
    float3 v1 = make_float3(from.x - origin.x, from.y - origin.y, from.z - origin.z);
    float3 v2 = make_float3(to.x - origin.x, to.y - origin.y, to.z - origin.z);
    
    // Normalize
    float len1 = sqrtf(v1.x*v1.x + v1.y*v1.y + v1.z*v1.z) + 1e-6f;
    float len2 = sqrtf(v2.x*v2.x + v2.y*v2.y + v2.z*v2.z) + 1e-6f;
    v1.x /= len1; v1.y /= len1; v1.z /= len1;
    v2.x /= len2; v2.y /= len2; v2.z /= len2;
    
    // Cross product
    float3 cross = make_float3(
        v1.y*v2.z - v1.z*v2.y,
        v1.z*v2.x - v1.x*v2.z,
        v1.x*v2.y - v1.y*v2.x
    );
    
    // Dot product
    float dot = v1.x*v2.x + v1.y*v2.y + v1.z*v2.z;
    dot = fminf(fmaxf(dot, -1.0f), 1.0f);
    
    // Angle magnitude
    float cross_mag = sqrtf(cross.x*cross.x + cross.y*cross.y + cross.z*cross.z);
    float angle = atan2f(cross_mag, dot);
    
    // Determine sign using face normal
    float normal_dot = cross.x*face_normal.x + cross.y*face_normal.y + cross.z*face_normal.z;
    if (normal_dot < 0) {
        angle = 2.0f * M_PI - angle;  // Make it the "other direction"
    }
    
    return angle;
}



__global__ void approximate_articulation_points_v1(
    const uint32_t* labels,  // Cluster Labels
    const bool*     border,
    const uint32_t* csr_edges,           // 1-hop neighbor data
    const uint32_t* csr_ptr,             // CSR pointers
    bool* is_simple_point,                // Output: true if simple point
    uint8_t* num_neq, // Output: Num p
    uint8_t* gcolors,
    uint8_t gchrome,
    uint32_t V                   // Number of vertices
) {
    

    // Warp and thread identification
    uint32_t vertex = (blockIdx.x * blockDim.x + threadIdx.x);
    if (vertex >= V) return;

    uint32_t my_label = labels[vertex];
    uint8_t gcolor = gcolors[vertex];
    if (gcolor != gchrome){ return; }
    if (!border[vertex]) { return; }

    uint32_t start = csr_ptr[vertex];
    uint32_t end = csr_ptr[vertex+1];
    // if ((end - start) > 3){
    //     printf("vertex[%d]: # of edges %d %d\n",vertex,start,end);
    // }
    //assert( (end - start) <= 3);
    //bool any_neq = false;
    uint8_t num = 0;
    uint8_t num_neq_v = 0;
    uint8_t num_eq_v = 0;
    for(int index = start; index < end; index++){
        uint32_t neigh = csr_edges[index];
        uint32_t neigh_vertex = labels[neigh];
        bool eq = neigh_vertex == my_label;
        num_neq_v += !eq;
        num_eq_v += eq;
        num = num + 1;
    }

    num_neq[vertex] = num_neq_v;
    float comp = max(1.0,0.2*(num));
    bool cond_b = (num_eq_v == 1) && (num >= 2); // retract the snakes
    is_simple_point[vertex] = (num_neq_v <= comp) || cond_b;
    
 
}


__global__ void approximate_articulation_points_v2( // travel over face
    const uint32_t* labels,  // Cluster Labels
    const bool*     border,
    const float3* pos,
    const uint32_t* csr_edges,           // 1-hop neighbor data
    const uint32_t* csr_ptr,             // CSR pointers
    bool* is_simple_point,                // Output: true if simple point
    uint8_t* num_neq, // Output: Num p
    uint8_t* gcolors,
    uint8_t gchrome,
    uint32_t V                   // Number of vertices
) {
    

    // Warp and thread identification
    uint32_t vertex = (blockIdx.x * blockDim.x + threadIdx.x);
    if (vertex >= V) return;

    uint32_t my_label = labels[vertex];
    uint8_t gcolor = gcolors[vertex];
    if (gcolor != gchrome){ return; }
    if (!border[vertex]) { return; }

    uint32_t start = csr_ptr[vertex];
    uint32_t end = csr_ptr[vertex+1];
    // if ((end - start) > 3){
    //     printf("vertex[%d]: # of edges %d %d\n",vertex,start,end);
    // }
    assert( (end - start) <= 3);
    //bool any_neq = false;
    uint8_t num_eq = 0;
    uint8_t num_neq_v = 0;
    float3 pos0;
    float3 pos1;
    uint32_t pos_neigh[2];
    for(int index = start; index < end; index++){
        uint32_t neigh = csr_edges[index];
        uint32_t neigh_vertex = labels[neigh];
        bool neq = neigh_vertex != my_label;
        num_neq_v += neq;
        if (!neq){
            if(num_eq == 0){
                pos0 = pos[neigh];
                pos_neigh[0] = neigh;
            }else if(num_eq == 1){
                pos1 = pos[neigh];
                pos_neigh[1] = neigh;
            }
            num_eq = num_eq + 1;
        }
    }

    num_neq[vertex] = num_neq_v;
    if (num_eq != 2){ 
        is_simple_point[vertex] = 0;
        return; 
    }
    // is_simple_point[vertex] = (num_neq_v == 1;

    // -- traverse face init --
    uint8_t MAX_STEPS = 32;
    uint8_t step = 0;
    float3 pos_v = pos[vertex];

    // -- get face normal --
    float3 face_normal;
    {
        // Face normal from cross product
        float3 v1 = make_float3(pos0.x - pos_v.x, pos0.y - pos_v.y, pos0.z - pos_v.z);
        float3 v2 = make_float3(pos1.x - pos_v.x, pos1.y - pos_v.y, pos1.z - pos_v.z);
        face_normal = make_float3(
            v1.y*v2.z - v1.z*v2.y,
            v1.z*v2.x - v1.x*v2.z,
            v1.x*v2.y - v1.y*v2.x
        );

        // Normalize it
        float len = sqrtf(face_normal.x*face_normal.x + face_normal.y*face_normal.y + face_normal.z*face_normal.z);
        face_normal.x /= len; face_normal.y /= len; face_normal.z /= len;
    }
    
    // -- .. --
    float angle = get_signed_angle(pos_v,pos0,pos1,face_normal);
    assert(angle < M_PI);

    // -- .. --
    float3 curr_pos = pos1; // start from pos1
    float3 prev_pos = pos_v; // start from pos1
    uint32_t curr_neigh = pos_neigh[1];
    uint32_t prev_neigh = vertex;

    while(curr_neigh != pos_neigh[0]){

        uint32_t start = csr_ptr[curr_neigh];
        uint32_t end = csr_ptr[curr_neigh+1];

        float curr_angle = 100000.f;
        uint32_t next_neigh = UINT32_MAX;
        float3 next_pos = make_float3(0,0,0);

        for(int index = start; index < end; index++){
            uint32_t neigh = csr_edges[index];
            if (neigh == prev_neigh) { continue; } // don't go back

            uint32_t neigh_label = labels[neigh];
            if(neigh_label != my_label){ continue; } // only stay in same label
            
            next_pos = pos[neigh];
            angle = get_signed_angle(curr_pos,prev_pos,next_pos,face_normal);
            if (angle < curr_angle){
                next_neigh = neigh;
            }
        }

        // -- terminate if failed to find next neighbor --
        if (next_neigh == UINT32_MAX){
            is_simple_point[vertex] = 0;
            return;
        }

        // -- update --
        prev_pos = curr_pos;
        prev_neigh = curr_neigh;
        curr_pos = pos[next_neigh];
        curr_neigh = next_neigh;

        // -- safety --
        step+=1;
        if(step > MAX_STEPS){
            is_simple_point[vertex] = 0;
            return;
        }
    }

    //printf("simple!\n");
    is_simple_point[vertex] = 1;
}


__global__ void approximate_articulation_points_v0(
    const uint32_t* labels,  // Cluster Labels
    const uint32_t* csr_edges,           // 1-hop neighbor data
    const uint32_t* csr_ptr,             // CSR pointers
    bool* is_simple_point,                // Output: true if simple point
    uint8_t* num_neq, // Output: Num p
    uint32_t V                   // Number of vertices
) {
    
    //  Warp Sizing
    constexpr uint32_t NUM_WARPS = 8;
    constexpr uint32_t WARP_SIZE = 32;
    constexpr uint8_t MAX_NEIGH = 32; // just happens to be 32; could be anything; 32 happens to also be # threads in warp.

    // Warp and thread identification
    uint32_t global_warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    int warp_in_block = threadIdx.x / WARP_SIZE;        // 0-7 (8 warps per block)
    int lane_id = threadIdx.x % WARP_SIZE;
    assert(warp_in_block < NUM_WARPS);
    
    if (global_warp_id >= V) return;
    
    //uint32_t vertex = candidates[global_warp_id];
    uint32_t vertex = global_warp_id;
    uint32_t my_label = labels[vertex];
    
    // Shared memory: 8 warps × 32 neighbors = 1KB per block
    __shared__ uint32_t shared_neighbors[NUM_WARPS][MAX_NEIGH];
    __shared__ uint32_t shared_labels[NUM_WARPS][MAX_NEIGH];
    __shared__ int shared_num_neighbors[NUM_WARPS];      // One per warp
    
    // Step 1: Cooperatively load 1-hop neighbors (coalesced)
    int start = csr_ptr[vertex];
    uint32_t num_neighbors = csr_ptr[vertex + 1] - start;
    shared_num_neighbors[warp_in_block] = num_neighbors;
    
    // Read Read Read
    for (int offset = 0; offset < MAX_NEIGH; offset += WARP_SIZE) {
        int idx = offset + lane_id;
        if (idx < num_neighbors && idx < MAX_NEIGH) {
            uint32_t neighbor = csr_edges[start + idx];
            shared_neighbors[warp_in_block][idx] = neighbor;
            shared_labels[warp_in_block][idx] = labels[neighbor];  // Direct access
        }
    }

    __syncwarp();

    // Count neighbors with different cluster labels
    int num_different_label = 0;
    for (int i = 0; i < num_neighbors && i < MAX_NEIGH; i++) {
        if (shared_labels[warp_in_block][i] != my_label) {
            num_different_label++;
        }
    }

    // Store result using lane 0
    if (lane_id == 0) {
        num_neq[global_warp_id] = num_different_label;
    }
    
    // Early exit for trivial cases
    if (num_neighbors <= 1) {
        if (lane_id == 0) {
            is_simple_point[global_warp_id] = true;  // Definitely simple
        }
        return;
    }
    
    // Handle vertices with too many neighbors
    if (num_neighbors >= MAX_NEIGH) {
        if (lane_id == 0) {
            is_simple_point[global_warp_id] = false;  // Conservative: assume not simple
        }
        return;
    }
    
    // Step 2: Check all neighbor pairs for 2-hop connectivity
    int total_pairs = (num_neighbors * (num_neighbors - 1)) / 2;
    bool thread_found_disconnection = false;
    
    // Each thread handles subset of pairs
    int pair_id = lane_id;
    while (pair_id < total_pairs && !thread_found_disconnection) {
        // Convert linear pair_id to (i,j) indices where i < j
        int temp_pair = pair_id;
        
        // Go from (pair_id) -> (i,j) neighbors
        // int i = (int)((sqrtf(1 + 8.0f * pair_id) - 1) / 2);
        // int j = pair_id - (i * (i + 1)) / 2 + i + 1;
        int i,j;
        upper_strict_from_index(num_neighbors, pair_id, i, j);
        
        if (i < num_neighbors && j < num_neighbors) {
            uint32_t n1 = shared_neighbors[warp_in_block][i];
            uint32_t n2 = shared_neighbors[warp_in_block][j];
            uint32_t l1 = shared_labels[warp_in_block][i];
            uint32_t l2 = shared_labels[warp_in_block][j];

            // Skip if either label isn't in the same cluster;
            // if ((l1 != my_label) || (l2 != my_label)){ continue; }

            // Check 2-hop connectivity
            bool connected;
            if ((l1 == my_label) && (l2 == my_label)){
                connected = check_2hop_connectivity(
                    n1, n2, vertex, 
                    csr_edges, csr_ptr,
                    shared_neighbors[warp_in_block], 
                    num_neighbors);
            }
            // }else{
            //     connected = false;
            // }
            
            if (!connected) {
                thread_found_disconnection = true;
            }
        }
        
        pair_id += WARP_SIZE;  // Next pair for this thread
    }
    
    // Step 3: Warp-level reduction
    bool warp_has_disconnection = __any_sync(0xFFFFFFFF, thread_found_disconnection);
    
    // Write result (simple point if no disconnections found)
    if (lane_id == 0) {
        is_simple_point[global_warp_id] = !warp_has_disconnection;
    }
}
