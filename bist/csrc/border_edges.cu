#include "hip/hip_runtime.h"



/******************************************************************
 * 
 *  Extract only the edges along the border so they can be 
 *  colored and thickened for neat visualizations.
 * 
 ********************************************************************/


#include "border_edges.h"
#include <hipcub/hipcub.hpp>


#define THREADS_PER_BLOCK 512


__global__ void
mark_edges(bool* marks, const uint32_t*spix, const bool* border, const uint32_t* edges, const uint32_t E){
	uint32_t edge_index = threadIdx.x + blockIdx.x * blockDim.x;  // the label
	if (edge_index>=E) return;
    uint32_t vertex0 = edges[2*edge_index];
    uint32_t vertex1 = edges[2*edge_index+1];
    bool both_edges = border[vertex0] && border[vertex1] && (spix[vertex0] == spix[vertex1]);
    marks[2*edge_index+0] = both_edges;
    marks[2*edge_index+1] = both_edges;
}


__global__ void
edge_counts(uint32_t* counts, const uint32_t* edges, const uint32_t E){
	uint32_t edge_index = threadIdx.x + blockIdx.x * blockDim.x;  // the label
	if (edge_index>=E) return;
    uint32_t vertex0 = edges[2*edge_index];
    //uint32_t vertex1 = edges[2*edge_index+1];
    atomicAdd(&counts[vertex0],1);
}


std::tuple<thrust::device_vector<uint32_t>,thrust::device_vector<uint32_t>>
get_border_edges(uint32_t* spix, bool* border, uint32_t* edges, uint32_t E){ //todo: enable batching.
    
    //
    // Part 1: Mark Edges to Keep
    //

    // -- launch parameters --
    int NumThreads = THREADS_PER_BLOCK;
    int edge_nblocks = ceil( double(E) / double(NumThreads) ); 
    dim3 EdgeBlocks(edge_nblocks);

    // -- mark edges to keep --
    thrust::device_vector<bool> marked(2*E, 0);
    bool* marked_ptr = thrust::raw_pointer_cast(marked.data());
    mark_edges<<<EdgeBlocks,NumThreads>>>(marked_ptr,spix,border,edges,E);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    //
    // Part 2: Shift Edges Down
    // 

    // -- create copy of edges --
    thrust::device_vector<uint32_t> border_edges(edges, edges + 2*E);

    // -- allocate memory --
    void* d_temp = nullptr;
    size_t temp_bytes = 0;
    unsigned int* d_num_selected;
    hipMalloc(&d_num_selected, sizeof(unsigned int));
    hipMemset(d_num_selected, 0, sizeof(unsigned int));

    // -- determine temp storage size --
    hipcub::DeviceSelect::Flagged(
        d_temp, temp_bytes,
        thrust::raw_pointer_cast(border_edges.data()),
        marked_ptr,d_num_selected,2*E
    );
    hipMalloc(&d_temp, temp_bytes);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // -- run --
    hipcub::DeviceSelect::Flagged(
        d_temp, temp_bytes,
        thrust::raw_pointer_cast(border_edges.data()),
        marked_ptr,d_num_selected,2*E
    );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // -- read num to keep --
    unsigned int nedges_twice;
    hipMemcpy(&nedges_twice,d_num_selected,sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipFree(d_num_selected);
    hipFree(d_temp);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // -- resize to filtered data -- 
    unsigned int nedges = nedges_twice/2;
    border_edges.resize(2*nedges);

    //printf("[border only shrinks it!] E,nedges_twice: %d %d\n",E,nedges_twice);
    //
    // -- Part 3: Read # Unique Edges and Reformat Output --
    //
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // -- get counts --
    // thrust::device_vector<uint32_t> counts(nedges,0);
    // int BorderEdgeBlocks = ceil( double(nedges) / double(NumThreads) ); 
    // edge_counts<<<BorderEdgeBlocks,NumThreads>>>(thrust::raw_pointer_cast(counts.data()),
    //                                              thrust::raw_pointer_cast(border_edges.data()),nedges);

    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );

    // -- get pointer --
    // thrust::device_vector<uint32_t> ptr(nedges+1,0);
    // thrust::inclusive_scan(counts.begin(), counts.end(), ptr.begin() + 1);
    thrust::device_vector<uint32_t> ptr; //  should be batchsize+1

    // -- view --
    // thrust::host_vector<uint32_t> bedges = border_edges;
    // for (int ix=0; ix < 10; ix++){
    //     printf("bedges[%d] = %d\n",ix,bedges[ix]);
    // }

    return {border_edges,ptr};
}


